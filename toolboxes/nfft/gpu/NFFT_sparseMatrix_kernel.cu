#include "hip/hip_runtime.h"
#include "vector_td.h"
#include "vector_td_utilities.h"
#include "setup_grid.h"
#include "cuSparseMatrix.h"
#include <thrust/fill.h>
namespace Gadgetron {

template<unsigned int N> struct iteration_counter{};

template<class REAL, unsigned int D, unsigned int N> __device__ __inline__ REAL ndim_loop(const vector_td<REAL,D> point, unsigned int & loop_counter, complext<REAL> * __restrict__ weights,
		int * __restrict__ column_indices,
		vector_td<int, D> & grid_point, const vector_td<int,D> & image_dims, const REAL W, const vector_td<REAL,D> & beta, iteration_counter<N>){
	REAL wsum = 0;
	for ( int i = ::ceil(point[N]-W*0.5); i <= ::floor(point[N]+W*0.5); i++){
		grid_point[N] = i;
		wsum += ndim_loop(point,loop_counter,weights,column_indices,grid_point,image_dims,W,beta,iteration_counter<N-1>());
	}
	return wsum;


}
template<class REAL, unsigned int D> __device__ __inline__ REAL ndim_loop(const vector_td<REAL,D> point, unsigned int & loop_counter, complext<REAL> * __restrict__ weights,
		int * __restrict__ column_indices,
		vector_td<int, D> & grid_point, const vector_td<int,D> & image_dims, const REAL W, const vector_td<REAL,D> & beta, iteration_counter<0>){

	REAL wsum =0;
	for ( int i = ::ceil(point[0]-W*0.5); i <= ::floor(point[0]+W*0.5); i++){
		grid_point[0] = i;
		REAL weight = KaiserBessel<REAL>(abs(point-vector_td<REAL,D>(grid_point)),vector_td<REAL,D>(image_dims),REAL(1)/W,beta);
		weights[loop_counter] = weight;
		//column_indices[loop_counter] = co_to_idx(grid_point%image_dims,image_dims);
		column_indices[loop_counter] = co_to_idx(grid_point%image_dims,image_dims);
		loop_counter++;
		wsum += weight;
	}
	return wsum;


}

template<class T> __device__ void index_sort(T* values, int* indices, int nvals){

	//Insertion sort, as we anticipate stuff to be mostly sorted. Might be faster to just sort the entire thing in a separate kernel
	for (int i = 0; i < nvals; i++){
		int index = indices[i];
		T val = values[i];
		int j = i;
		while (j > 0 && indices[j-1] > index){
			indices[j] = indices[j-1];
			values[j] = values[j-1];
			j--;
		}
		indices[j] = index;
		values[j] = val;

	}

}

/**
 *
 * @param points non-cartesian points on which to do the NFFT. Size tot_size
 * @param offsets Array containing the offsets at which the rows should be stored. Size tot_size+1
 * @param weights Output array which will contain the values of the sparse matrix.
 * @param column_indices Output array containing the column indices of the sparse matrix
 * @param tot_size Total number of points
 */
template<class REAL, unsigned int D> __global__ void make_NFFT_matrix_kernel(const vector_td<REAL,D> * __restrict__ points, const int * __restrict__ offsets, complext<REAL> * __restrict__ weights, int * __restrict__ column_indices, const vector_td<int,D> image_dims, const vector_td<REAL,D> beta, const REAL W, unsigned int tot_size ){
	const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < tot_size){
		vector_td<REAL,D> p = points[idx];
		const int offset = offsets[idx];


		complext<REAL> * local_weight = weights+offset;
		int * local_column_indices = column_indices+offset;
		unsigned int loop_counter = 0;
		vector_td<int,D> grid_point;
		REAL wsum = ndim_loop(p,loop_counter, local_weight,local_column_indices, grid_point,image_dims,W,beta,iteration_counter<D-1>());
		REAL inv_wsum = 1.0/wsum;
		for (unsigned int i = offset; i < offsets[idx+1]; i++){
			weights[i] *= inv_wsum;
		}
		index_sort(local_weight,local_column_indices,offsets[idx+1]-offset);

	}
}

template<class REAL> void check_csrMatrix(cuCsrMatrix<complext<REAL> > &matrix){

	if (matrix.csrRow.size() != matrix.m+1){
		throw std::runtime_error("Malformed CSR matrix: length of CSR vector does not match matrix size");
	}

	if (matrix.csrColdnd.size() != matrix.nnz){
		throw std::runtime_error("Malformed CSR matrix: length of column indices vector does not match number of non-zero elements");
	}
	if (matrix.data.size() != matrix.nnz ){
		throw std::runtime_error("Malformed CSR matrix: length of data vector does not match number of non-zero elements");
	}

	int min_ind = *thrust::min_element(matrix.csrColdnd.begin(),matrix.csrColdnd.end());
	int max_ind = *thrust::max_element(matrix.csrColdnd.begin(),matrix.csrColdnd.end());

	if (min_ind < 0 || max_ind > matrix.n){
		std::stringstream ss;
		ss << "Malformed CSR matrix: column indices vector contains illegal values. Min " << min_ind<< " max " << max_ind;
		throw std::runtime_error(ss.str());
	}
	int min_row = *thrust::min_element(matrix.csrRow.begin(),matrix.csrRow.end());
	int max_row = *thrust::max_element(matrix.csrRow.begin(),matrix.csrRow.end());

	if (min_row < 0 || max_row != matrix.nnz){
		throw std::runtime_error("Malformed CSR matrix: CSR vector conains illegal values");
	}


	if (isnan(abs(thrust::reduce(matrix.data.begin(),matrix.data.end()))))
		throw std::runtime_error("Matrix contains NaN");


}

template<class REAL, unsigned int D> boost::shared_ptr<cuCsrMatrix<complext<REAL> > > make_NFFT_matrix( thrust::device_vector<vector_td<REAL,D> > & points,  const vector_td<size_t,D> image_dims, const vector_td<REAL,D> beta, const REAL W ){

	boost::shared_ptr<cuCsrMatrix<complext<REAL> > > matrix(new cuCsrMatrix<complext<REAL> >);

	matrix->csrRow = thrust::device_vector<int>(points.size()+1);
	matrix->csrRow[0] = 0;
	CHECK_FOR_CUDA_ERROR();

	REAL half_W = REAL(0.5)*W;
	{
		thrust::device_vector<int> c_p_s(points.size());
		thrust::transform(points.begin(), points.end(), c_p_s.begin(), compute_num_cells_per_sample<REAL,D>(half_W));

		thrust::inclusive_scan( c_p_s.begin(), c_p_s.end(), matrix->csrRow.begin()+1, thrust::plus<int>()); // prefix sum


	}
	unsigned int num_pairs = matrix->csrRow.back();
	//cuNDArray<int> row_indices(ind_dims);
	matrix->csrColdnd = thrust::device_vector<int>(num_pairs);
	matrix->data = thrust::device_vector<complext<REAL> >(num_pairs);
	//cuNDArray<complext<REAL> > values(ind_dims);


	dim3 dimBlock;
	dim3 dimGrid;
	setup_grid(points.size(),&dimBlock,&dimGrid);

	make_NFFT_matrix_kernel<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(&points[0]),thrust::raw_pointer_cast(&matrix->csrRow[0]), thrust::raw_pointer_cast(&matrix->data[0]), thrust::raw_pointer_cast(&matrix->csrColdnd[0]),vector_td<int,D>(image_dims),beta,W, points.size() );
	hipDeviceSynchronize();
	CHECK_FOR_CUDA_ERROR();
	matrix->m = points.size();
	matrix->n = prod(image_dims);
	matrix->nnz = num_pairs;

 std::cout << " Matrix sum: " << thrust::reduce(matrix->data.begin(),matrix->data.end()) << std::endl;
	//cusparseSet
	//CUSPARSE_CALL(hipsparseSetMatType(matrix->descr,HIPSPARSE_MATRIX_TYPE_GENERAL));
	//CUSPARSE_CALL(hipsparseSetMatDiagType(matrix->descr,HIPSPARSE_DIAG_TYPE_NON_UNIT));
	//CUSPARSE_CALL(hipsparseSetMatIndexBase(matrix->descr,HIPSPARSE_INDEX_BASE_ZERO));

	return matrix;
}

}


