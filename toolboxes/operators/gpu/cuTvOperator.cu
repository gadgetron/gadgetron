#include "hip/hip_runtime.h"
#include "cuTvOperator.h"
#include "cuNDArray_operators.h"
#include "cuNDArray_elemwise.h"
#include "vector_td_utilities.h"
#include "complext.h"
#include <iostream>
#include "check_CUDA.h"
#include "cudaDeviceManager.h"
#include <stdio.h>

using namespace Gadgetron;

template<class REAL, class T, unsigned int D> static inline  __device__ REAL gradient(const T* __restrict__ in, const vector_td<int,D>& dims, vector_td<int,D>& co)
{
	REAL grad = REAL(0);
	T xi = in[co_to_idx((co+dims)%dims,dims)];
	for (int i = 0; i < D; i++){
		co[i]+=1;
		T dt = in[co_to_idx((co+dims)%dims,dims)];
		grad += norm(xi-dt);
		co[i]-=1;
	}
	return sqrt(grad);
}


template<class REAL, class T, unsigned int D> static __global__ void tvGradient_kernel(const T* __restrict__ in, T* __restrict__ out, const vector_td<int,D> dims,REAL limit,REAL weight)
{
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	if( idx < prod(dims) ){
		T xi = in[idx];
		T result=T(0);

		vector_td<int,D> co = idx_to_co(idx, dims);

		REAL grad = ::max(gradient<REAL,T,D>(in,dims,co),limit);

		if (grad > limit) {
			//result += REAL(D)*xi/grad;
			for (int i = 0; i < D; i++){
				co[i]+=1;
				result += (xi-in[co_to_idx((co+dims)%dims,dims)])/grad;
				co[i]-=1;
			}
		}

		for (int i = 0; i < D; i++){
			co[i]-=1;
			grad = ::max(gradient<REAL,T,D>(in,dims,co),limit);

			if (grad > limit) {
				result +=(xi-in[co_to_idx((co+dims)%dims,dims)])/grad;
			}
			co[i]+=1;
		}
		out[idx] += result*weight;
	}
}








template<class T, unsigned int D> void cuTvOperator<T,D>::gradient (cuNDArray<T> * in,cuNDArray<T> * out, bool accumulate)
{
	if (!accumulate)
		clear(out);

	const typename intd<D>::Type dims = vector_td<int,D>( from_std_vector<size_t,D>(in->get_dimensions()));
	int elements = in->get_number_of_elements();

	int threadsPerBlock =std::min(prod(dims),256); //Using hardcoded blockSize because we use quite a lot of registers

	dim3 dimBlock( threadsPerBlock);
	int totalBlocksPerGridx = std::min(std::max(1,prod(dims)/threadsPerBlock),cudaDeviceManager::Instance()->max_griddim());
	int totalBlocksPerGridy = (prod(dims)-1)/(threadsPerBlock*totalBlocksPerGridx)+1;
	dim3 dimGrid(totalBlocksPerGridx,totalBlocksPerGridy);

	for (int i =0; i < (elements/prod(dims)); i++){
		tvGradient_kernel<<<dimGrid,dimBlock>>>(in->get_data_ptr()+i*prod(dims),out->get_data_ptr()+i*prod(dims),dims,limit_,this->weight_);
	}

	//hipDeviceSynchronize();
	//CHECK_FOR_CUDA_ERROR();
}

template<class REAL, class T, unsigned int D> static __global__ void tvMagnitude_kernel(const  T* __restrict__  in,T* __restrict__ out,const vector_td<int,D> dims,REAL limit,REAL weight)
{
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	if( idx < prod(dims) ){
		vector_td<int,D> co = idx_to_co(idx, dims);
		REAL grad = gradient<REAL,T,D>(in,dims,co);
		//out[idx] =  (grad > limit) ? grad*weight : REAL(0);
		out[idx] = grad*weight;
	}
}


template<class T, unsigned int D> typename realType<T>::Type cuTvOperator<T,D>::magnitude (cuNDArray<T> * in)
{
	cuNDArray<T> out(in->get_dimensions());
	const typename intd<D>::Type dims = vector_td<int,D>( from_std_vector<size_t,D>(in->get_dimensions()));
	int elements = in->get_number_of_elements();

	int threadsPerBlock =std::min(prod(dims),256); //Using hardcoded blockSize because we use quite a lot of registers
	dim3 dimBlock( threadsPerBlock);
	int totalBlocksPerGridx = std::min(std::max(1,prod(dims)/threadsPerBlock),cudaDeviceManager::Instance()->max_griddim());
	int totalBlocksPerGridy = (prod(dims)-1)/(threadsPerBlock*totalBlocksPerGridx)+1;
	dim3 dimGrid(totalBlocksPerGridx,totalBlocksPerGridy);

	for (int i =0; i < (elements/prod(dims)); i++){
		tvMagnitude_kernel<<<dimGrid,dimBlock>>>(in->get_data_ptr()+i*prod(dims),out.get_data_ptr()+i*prod(dims),dims,limit_,this->weight_);
	}

	//hipDeviceSynchronize();
	//CHECK_FOR_CUDA_ERROR();
	return asum(&out);
}

template class cuTvOperator<float,1>;
template class cuTvOperator<float,2>;
template class cuTvOperator<float,3>;
template class cuTvOperator<float,4>;

template class cuTvOperator<double,1>;
template class cuTvOperator<double,2>;
template class cuTvOperator<double,3>;
template class cuTvOperator<double,4>;

template class cuTvOperator<float_complext,1>;
template class cuTvOperator<float_complext,2>;
template class cuTvOperator<float_complext,3>;
template class cuTvOperator<float_complext,4>;

template class cuTvOperator<double_complext,1>;
template class cuTvOperator<double_complext,2>;
template class cuTvOperator<double_complext,3>;
template class cuTvOperator<double_complext,4>;
