#include "hip/hip_runtime.h"
#include "cuConvolutionOperator.h"
#include "vector_td_utilities.h"
#include "cudaDeviceManager.h"
#include "setup_grid.h"

namespace Gadgetron {

  // Mirror, but keep the origin unchanged
  template<class T, unsigned int D> __global__ void
  origin_mirror_kernel( vector_td<unsigned int,D> matrix_size, vector_td<unsigned int,D> origin, const T * __restrict__ in, T * __restrict__ out, bool zero_fill )
  {
    const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    
    if( idx < prod(matrix_size) ){
      
      vector_td<unsigned int,D> in_co = idx_to_co( idx, matrix_size );
      vector_td<unsigned int,D> out_co = matrix_size-in_co;
    
      bool wrap = false;
      for( unsigned int d=0; d<D; d++ ){
	if( out_co.vec[d] == matrix_size.vec[d] ){
	  out_co.vec[d] = 0;
	  wrap = true;
	}
      }
    
      const unsigned int in_idx = co_to_idx(in_co, matrix_size);
      const unsigned int out_idx = co_to_idx(out_co, matrix_size);

      if( wrap && zero_fill )
	out[out_idx] = T(0);
      else
	out[out_idx] = in[in_idx];
    }
  }
  
  // Mirror around the origin -- !! leaving the origin unchanged !!
  // This creates empty space "on the left" that can be filled by zero (default) or the left-over entry.
  template<class REAL, unsigned int D> void
  cuConvolutionOperator<REAL,D>::origin_mirror( cuNDArray< complext<REAL> > *in, cuNDArray< complext<REAL> > *out )
  {
    if( in == 0x0 || out == 0x0 ){
      throw std::runtime_error( "origin_mirror: 0x0 ndarray provided");
    }
    
    if( !in->dimensions_equal(out) ){
      throw std::runtime_error("origin_mirror: image dimensions mismatch");
    }
    
    if( in->get_number_of_dimensions() != D ){
      std::stringstream ss;
      ss << "origin_mirror: number of image dimensions is not " << D;
      throw std::runtime_error(ss.str());
    }

    typename uint64d<D>::Type matrix_size = from_std_vector<size_t,D>( in->get_dimensions() );
  
    // Setup block/grid dimensions
    dim3 blockDim; dim3 gridDim;
    setup_grid( prod(matrix_size), &blockDim, &gridDim );

    // Invoke kernel
    origin_mirror_kernel<complext<REAL>,D><<< gridDim, blockDim >>> 
      ( vector_td<unsigned int,D>(matrix_size), vector_td<unsigned int,D>(matrix_size>>1), in->get_data_ptr(), out->get_data_ptr(), true );
    
    CHECK_FOR_CUDA_ERROR();
  }


  template <class REAL, unsigned int D> void 
  cuConvolutionOperator<REAL,D>::operator_fft( bool forwards_transform, cuNDArray< complext<REAL> > *image )
  {
    if( forwards_transform )
      cuNDFFT<REAL>::instance()->fft(image);
    else
      cuNDFFT<REAL>::instance()->ifft(image);
  }    
  
  template EXPORTGPUOPERATORS class cuConvolutionOperator<float,1>;
  template EXPORTGPUOPERATORS class cuConvolutionOperator<float,2>;
  template EXPORTGPUOPERATORS class cuConvolutionOperator<float,3>;
  template EXPORTGPUOPERATORS class cuConvolutionOperator<float,4>;

  template EXPORTGPUOPERATORS class cuConvolutionOperator<double,1>;
  template EXPORTGPUOPERATORS class cuConvolutionOperator<double,2>;
  template EXPORTGPUOPERATORS class cuConvolutionOperator<double,3>;
  template EXPORTGPUOPERATORS class cuConvolutionOperator<double,4>;
  
}
