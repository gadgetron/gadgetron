#include "hip/hip_runtime.h"
#include "cuLaplaceOperator.h"
#include "cuNDArray_operators.h"
#include "cuNDArray_elemwise.h"
#include "vector_td.h"
#include "vector_td_utilities.h"
#include "check_CUDA.h"

namespace Gadgetron{

  // Template Power function
  template<unsigned int i, unsigned int j>
  struct Pow
  {
    enum { Value = i*Pow<i,j-1>::Value};
  };

  template <unsigned int i>
  struct Pow<i,1>
  {
    enum { Value = i};
  };

  template<class T, unsigned int D, unsigned int dim> class inner_laplace_functor{
  public:
		static __device__ __inline__ void apply(T& val,const T* __restrict__ in, const typename intd<D>::Type dims,const typename intd<D>::Type co, typename intd<D>::Type& stride){
			for (int d = -1; d < 2; d++)
				stride[dim]=d;
				inner_laplace_functor<T,D,dim-1>::apply(val,in,dims,co,stride);
		}
  };
  template<class T, unsigned int D> class inner_laplace_functor<T,D,0>{
  public:
  	static __device__ __inline__ void apply(T& val,const T* __restrict__ in, const typename intd<D>::Type dims,const typename intd<D>::Type co, typename intd<D>::Type& stride){
  		typename intd<D>::Type coN = (co+dims+stride)%dims;
  		val -= in[co_to_idx(coN,dims)];
  	}
  };

  template<class REAL, class T, unsigned int D> __global__ void
  laplace_kernel( typename intd<D>::Type dims, const T * __restrict__ in, T * __restrict__ out )
  {  
    const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    if( idx < prod(dims) ){
    
      T val = T(0);

      typename intd<D>::Type co = idx_to_co(idx, dims);
      typename intd<D>::Type stride(0);

      inner_laplace_functor<T,D,D-1>::apply(val,in,dims,co,stride);
      out[idx] = val+in[co_to_idx(co, dims)]*((REAL) Pow<3,D>::Value);
    }
  }

  template< class T, unsigned int D> void
  cuLaplaceOperator<T,D>::compute_laplace( cuNDArray<T> *in, cuNDArray<T> *out, bool accumulate )
  {
  
    if( !in || !out || in->get_number_of_elements() != out->get_number_of_elements() ){
      throw std::runtime_error("laplaceOperator::compute_laplace : array dimensions mismatch.");

    }
  
    typename intd<D>::Type dims = vector_td<int,D>( from_std_vector<size_t,D>( *(in->get_dimensions().get()) ));

    dim3 dimBlock( dims[0] );
    dim3 dimGrid( prod(dims)/dims[0] );
  
    // Invoke kernel
    laplace_kernel<typename realType<T>::Type ,T,D><<< dimGrid, dimBlock >>> (dims, in->get_data_ptr(), out->get_data_ptr() );
  
    CHECK_FOR_CUDA_ERROR();
  }
  
  // Instantiations

  template class EXPORTGPUOPERATORS cuLaplaceOperator<float, 1>;
  template class EXPORTGPUOPERATORS cuLaplaceOperator<float, 2>;
  template class EXPORTGPUOPERATORS cuLaplaceOperator<float, 3>;

  template class EXPORTGPUOPERATORS cuLaplaceOperator<float_complext, 1>;
  template class EXPORTGPUOPERATORS cuLaplaceOperator<float_complext, 2>;
  template class EXPORTGPUOPERATORS cuLaplaceOperator<float_complext, 3>;

  template class EXPORTGPUOPERATORS cuLaplaceOperator<double, 1>;
  template class EXPORTGPUOPERATORS cuLaplaceOperator<double, 2>;
  template class EXPORTGPUOPERATORS cuLaplaceOperator<double, 3>;

  template class EXPORTGPUOPERATORS cuLaplaceOperator<double_complext, 1>;
  template class EXPORTGPUOPERATORS cuLaplaceOperator<double_complext, 2>;
  template class EXPORTGPUOPERATORS cuLaplaceOperator<double_complext, 3>;
}
