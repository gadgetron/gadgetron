#include "hip/hip_runtime.h"
#include "cuTv1dOperator.h"
#include "cuNDArray_operators.h"
#include "cuNDArray_elemwise.h"
#include "vector_td_utilities.h"
#include "complext.h"
#include "check_CUDA.h"
#include "cudaDeviceManager.h"

#include <iostream>

using namespace Gadgetron;

template<class REAL, class T, unsigned int D> static inline	__device__ REAL gradient(const T* __restrict__ in, const vector_td<int,D>& dims, vector_td<int,D>& co){

	T xi = in[co_to_idx((co+dims)%dims,dims)];

	co[D-1]+=1;
	T dt = in[co_to_idx((co+dims)%dims,dims)];
	REAL grad = norm(xi-dt);
	co[D-1]-=1;

	return sqrt(grad);
}


template<class REAL, class T, unsigned int D> static __global__ void tvGradient_kernel(const T* __restrict__ in, T* __restrict__ out, const vector_td<int,D> dims,REAL limit,REAL weight){
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	if( idx < prod(dims) ){
		T xi = in[idx];
		T result=T(0);

		vector_td<int,D> co = idx_to_co(idx, dims);

		REAL grad = gradient<REAL,T,D>(in,dims,co);


		if (grad > limit) {
			result += xi/grad;

			co[D-1]+=1;
			result -= in[co_to_idx((co+dims)%dims,dims)]/grad;
			co[D-1]-=1;

		}

		co[D-1]-=1;
		grad = gradient<REAL,T,D>(in,dims,co);
		if (grad > limit) {
			result +=(xi-in[co_to_idx((co+dims)%dims,dims)])/grad;
		}
		co[D-1]+=1;

		out[idx] += weight*result;

	}
}


template<class T, unsigned int D> void cuTv1DOperator<T,D>::gradient (cuNDArray<T> * in,cuNDArray<T> * out, bool accumulate){
	if (!accumulate) clear(out);

	const typename intd<D>::Type dims = vector_td<int,D>( from_std_vector<size_t,D>(in->get_dimensions()));
	int elements = in->get_number_of_elements();

	int threadsPerBlock =std::min(prod(dims),cudaDeviceManager::Instance()->max_blockdim());
	dim3 dimBlock( threadsPerBlock);
	int totalBlocksPerGrid = std::max(1,prod(dims)/cudaDeviceManager::Instance()->max_blockdim());
	dim3 dimGrid(totalBlocksPerGrid);

	for (int i =0; i < (elements/prod(dims)); i++){
		tvGradient_kernel<<<dimGrid,dimBlock>>>(in->get_data_ptr()+i*prod(dims),out->get_data_ptr()+i*prod(dims),dims,limit_,this->weight_);
	}


	hipDeviceSynchronize();
	CHECK_FOR_CUDA_ERROR();
}

template<class REAL, class T, unsigned int D> static __global__ void tvMagnitude_kernel(const T* in,T* out,const vector_td<int,D> dims,REAL limit,REAL weight)
{
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	if( idx < prod(dims) ){
		vector_td<int,D> co = idx_to_co(idx, dims);
		REAL grad = gradient<REAL,T,D>(in,dims,co);
		out[idx] = grad*weight;
	}
}

template<class T, unsigned int D> typename realType<T>::Type cuTv1DOperator<T,D>::magnitude (cuNDArray<T> * in){

	cuNDArray<T> out(*in);
	const typename intd<D>::Type dims = vector_td<int,D>( from_std_vector<size_t,D>(in->get_dimensions()));
	int elements = in->get_number_of_elements();

	int threadsPerBlock =std::min(prod(dims),cudaDeviceManager::Instance()->max_blockdim());
	dim3 dimBlock( threadsPerBlock);
	int totalBlocksPerGrid = std::max(1,prod(dims)/cudaDeviceManager::Instance()->max_blockdim());
	dim3 dimGrid(totalBlocksPerGrid);

	for (int i =0; i < (elements/prod(dims)); i++){
		tvMagnitude_kernel<<<dimGrid,dimBlock>>>(in->get_data_ptr()+i*prod(dims),out.get_data_ptr()+i*prod(dims),dims,limit_,this->weight_);
	}


	hipDeviceSynchronize();
	CHECK_FOR_CUDA_ERROR();
	return asum(&out);
}


template class EXPORTGPUOPERATORS cuTv1DOperator<float,1>;
template class EXPORTGPUOPERATORS cuTv1DOperator<float,2>;
template class EXPORTGPUOPERATORS cuTv1DOperator<float,3>;
template class EXPORTGPUOPERATORS cuTv1DOperator<float,4>;

template class EXPORTGPUOPERATORS cuTv1DOperator<double,1>;
template class EXPORTGPUOPERATORS cuTv1DOperator<double,2>;
template class EXPORTGPUOPERATORS cuTv1DOperator<double,3>;
template class EXPORTGPUOPERATORS cuTv1DOperator<double,4>;

template class EXPORTGPUOPERATORS cuTv1DOperator<float_complext,1>;
template class EXPORTGPUOPERATORS cuTv1DOperator<float_complext,2>;
template class EXPORTGPUOPERATORS cuTv1DOperator<float_complext,3>;
template class EXPORTGPUOPERATORS cuTv1DOperator<float_complext,4>;

template class EXPORTGPUOPERATORS cuTv1DOperator<double_complext,1>;
template class EXPORTGPUOPERATORS cuTv1DOperator<double_complext,2>;
template class EXPORTGPUOPERATORS cuTv1DOperator<double_complext,3>;
template class EXPORTGPUOPERATORS cuTv1DOperator<double_complext,4>;
