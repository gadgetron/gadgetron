#include "hip/hip_runtime.h"
#include "sense_utilities.h"
#include "vector_td_utilities.h"
#include <sstream>

namespace Gadgetron{

  template<class REAL> __global__ void
  mult_csm_kernel( const complext<REAL> * __restrict__ in, complext<REAL> * __restrict__ out, complext<REAL> *csm,
		   size_t image_elements, unsigned int nframes, unsigned int ncoils )
  {
    unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if( idx < image_elements) {
      complext<REAL> _in = in[idx+blockIdx.y*image_elements];
      for( unsigned int i=0; i<ncoils; i++) {
	out[idx + blockIdx.y*image_elements + i*image_elements*nframes] =  _in * csm[idx+i*image_elements];
      }
    }
  }

  template<class REAL, unsigned int D> void
  csm_mult_M( cuNDArray< complext<REAL> > *in, cuNDArray< complext<REAL> > *out, cuNDArray< complext<REAL> > *csm )
  {
    int device;
    if( hipGetDevice( &device ) != hipSuccess ){
      throw cuda_error( "mult_csm: unable to query current device");
    }

    if( !in || in->get_device() != device || !out || out->get_device() != device || !csm || csm->get_device() != device ){
      throw cuda_error("mult_csm: array not residing current device");
    }

    if( in->get_number_of_dimensions() < D  || in->get_number_of_dimensions() > D+1 ){
      throw std::runtime_error("mult_csm: unexpected input dimensionality");
    }

    if( in->get_number_of_dimensions() > out->get_number_of_dimensions() ){
      throw std::runtime_error("mult_csm: input dimensionality cannot exceed output dimensionality");
    }

    if( csm->get_number_of_dimensions() != D+1 ) {
      throw std::runtime_error("mult_csm: input dimensionality of csm not as expected");
    }

    unsigned int num_image_elements = 1;
    for( unsigned int d=0; d<D; d++ )
      num_image_elements *= in->get_size(d);

    unsigned int num_frames = in->get_number_of_elements() / num_image_elements;

    dim3 blockDim(256);
    dim3 gridDim((num_image_elements+blockDim.x-1)/blockDim.x, num_frames);

    mult_csm_kernel<REAL><<< gridDim, blockDim >>>
      ( in->get_data_ptr(), out->get_data_ptr(), csm->get_data_ptr(), num_image_elements, num_frames, csm->get_size(D) );

    hipError_t err = hipGetLastError();
    if( err != hipSuccess ){
      std::stringstream ss;
      ss << "mult_csm: unable to multiply with coil sensitivities: " <<
	hipGetErrorString(err);
      throw cuda_error(ss.str());

    }
  }

  template <class REAL> __global__ void
  mult_csm_conj_sum_kernel(const  complext<REAL> * __restrict__ in, complext<REAL> * __restrict__ out, const complext<REAL> * __restrict__ csm,
			    size_t image_elements, unsigned int nframes, unsigned int ncoils )
  {
    unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if( idx < image_elements ) {
      complext<REAL> _out =complext<REAL>(0);
      for( unsigned int i = 0; i < ncoils; i++ ) {
	_out += in[idx+blockIdx.y*image_elements+i*nframes*image_elements] * conj(csm[idx+i*image_elements]);
      }
      out[idx+blockIdx.y*image_elements] = _out;
    }
  }

  template<class REAL, unsigned int D> void
  csm_mult_MH( cuNDArray<complext<REAL> > *in, cuNDArray<complext<REAL> > *out, cuNDArray<complext<REAL> > *csm )
  {
    int device;
    if( hipGetDevice( &device ) != hipSuccess ){
      throw cuda_error("mult_csm_conj_sum: unable to query current device");
    }

    if( !in || in->get_device() != device || !out || out->get_device() != device || !csm || csm->get_device() != device ){
      throw std::runtime_error("mult_csm_conj_sum: array not residing current device");
    }

    if( out->get_number_of_dimensions() < D  || out->get_number_of_dimensions() > D+1 ){
      throw std::runtime_error("mult_csm_conj_sum: unexpected output dimensionality");
    }

    if( out->get_number_of_dimensions() > in->get_number_of_dimensions() ){
      throw std::runtime_error("mult_csm_conj_sum: output dimensionality cannot exceed input dimensionality");
    }

    if( csm->get_number_of_dimensions() != D+1 ) {
      throw std::runtime_error("mult_csm_conj_sum: input dimensionality of csm not as expected");
    }

    unsigned int num_image_elements = 1;
    for( unsigned int d=0; d<D; d++ )
      num_image_elements *= out->get_size(d);

    unsigned int num_frames = out->get_number_of_elements() / num_image_elements;

    dim3 blockDim(256);
    dim3 gridDim((num_image_elements+blockDim.x-1)/blockDim.x, num_frames);

    mult_csm_conj_sum_kernel<REAL><<< gridDim, blockDim >>>
      ( in->get_data_ptr(), out->get_data_ptr(), csm->get_data_ptr(), num_image_elements, num_frames, csm->get_size(D) );

    hipError_t err = hipGetLastError();
    if( err != hipSuccess ){
      std::stringstream ss;
      ss << "mult_csm_conj_sum: unable to combine coils " <<
	hipGetErrorString(err);
      throw cuda_error(ss.str());
    }
  }

  // Instantiation

  template void csm_mult_M<float,1>( cuNDArray< complext<float> >*, cuNDArray< complext<float> >*, cuNDArray< complext<float> >*);
  template void csm_mult_M<float,2>( cuNDArray< complext<float> >*, cuNDArray< complext<float> >*, cuNDArray< complext<float> >*);
  template void csm_mult_M<float,3>( cuNDArray< complext<float> >*, cuNDArray< complext<float> >*, cuNDArray< complext<float> >*);
  template void csm_mult_M<float,4>( cuNDArray< complext<float> >*, cuNDArray< complext<float> >*, cuNDArray< complext<float> >*);

  template void csm_mult_M<double,1>( cuNDArray< complext<double> >*, cuNDArray< complext<double> >*, cuNDArray< complext<double> >*);
  template void csm_mult_M<double,2>( cuNDArray< complext<double> >*, cuNDArray< complext<double> >*, cuNDArray< complext<double> >*);
  template void csm_mult_M<double,3>( cuNDArray< complext<double> >*, cuNDArray< complext<double> >*, cuNDArray< complext<double> >*);
  template void csm_mult_M<double,4>( cuNDArray< complext<double> >*, cuNDArray< complext<double> >*, cuNDArray< complext<double> >*);

  template void csm_mult_MH<float,1>( cuNDArray< complext<float> >*, cuNDArray< complext<float> >*, cuNDArray< complext<float> >*);
  template void csm_mult_MH<float,2>( cuNDArray< complext<float> >*, cuNDArray< complext<float> >*, cuNDArray< complext<float> >*);
  template void csm_mult_MH<float,3>( cuNDArray< complext<float> >*, cuNDArray< complext<float> >*, cuNDArray< complext<float> >*);
  template void csm_mult_MH<float,4>( cuNDArray< complext<float> >*, cuNDArray< complext<float> >*, cuNDArray< complext<float> >*);

  template void csm_mult_MH<double,1>( cuNDArray< complext<double> >*, cuNDArray< complext<double> >*, cuNDArray< complext<double> >*);
  template void csm_mult_MH<double,2>( cuNDArray< complext<double> >*, cuNDArray< complext<double> >*, cuNDArray< complext<double> >*);
  template void csm_mult_MH<double,3>( cuNDArray< complext<double> >*, cuNDArray< complext<double> >*, cuNDArray< complext<double> >*);
  template void csm_mult_MH<double,4>( cuNDArray< complext<double> >*, cuNDArray< complext<double> >*, cuNDArray< complext<double> >*);
}
