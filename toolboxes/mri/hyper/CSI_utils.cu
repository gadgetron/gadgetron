#include "hip/hip_runtime.h"
#include "CSI_utils.h"
#include <algorithm>
#include "cudaDeviceManager.h"
#include "complext.h"
#include <hip/hip_math_constants.h>
#include <stdio.h>
using namespace Gadgetron;


template<class T> static __global__ void dft_kernel(complext<T>* __restrict__ kspace, const complext<T>* __restrict__ tspace, T* __restrict__ frequencies, unsigned int spiral_length, unsigned int echoes, unsigned int nfreqs,T dte, T dtt){
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < spiral_length*nfreqs ){
		complext<T> result = 0;
		T frequency = frequencies[idx/spiral_length];
		T time_offset = dtt*(idx%spiral_length);
		unsigned int kpoint = idx%spiral_length;
		for (unsigned int i =0; i < echoes; i++){
			result += exp(complext<T>(0,-frequency*2*HIP_PI_F*(dte*i+time_offset)))*tspace[kpoint+i*spiral_length];
		}
		kspace[idx] = result;
	}
}

template<class T> static __global__ void dftH_kernel(const complext<T>* __restrict__ kspace, complext<T>* __restrict__ tspace, T* __restrict__ frequencies, unsigned int spiral_length, unsigned int echoes, unsigned int nfreqs,T dte, T dtt){
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < spiral_length*echoes ){
		complext<T> result = 0;
		unsigned int kpoint = idx%spiral_length;
		T timeshift = dte*(idx/spiral_length)+dtt*kpoint;
		for (unsigned int i =0; i < nfreqs; i++){
			result += exp(complext<T>(0,frequencies[i]*2*HIP_PI_F*timeshift))*kspace[kpoint+i*spiral_length];
		}
		tspace[idx] = result;
	}
}



template<class T>
void Gadgetron::CSI_dft(cuNDArray<complext<T> >* kspace,
		cuNDArray<complext<T> >* tspace, thrust::device_vector<T>* frequencies, T dtt, T dte) {

	size_t elements = kspace->get_size(0)*kspace->get_size(1);
	size_t batches = kspace->get_number_of_elements()/elements;
	size_t t_elements = tspace->get_size(0)*tspace->get_size(1);
	for (int i = 0; i< batches; i++){
		int threadsPerBlock = std::min<int>(elements,cudaDeviceManager::Instance()->max_blockdim());
		dim3 dimBlock(threadsPerBlock);
		int totalBlocksPerGrid = (elements+threadsPerBlock-1)/threadsPerBlock;
		dim3 dimGrid(totalBlocksPerGrid);

		if (totalBlocksPerGrid > cudaDeviceManager::Instance()->max_griddim())
			throw std::runtime_error("CSIOperator: Input dimensions too large");

		//size_t batchSize = dimGrid.x*dimBlock.x;
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(dft_kernel<T>),hipFuncCachePreferL1);

		std::vector<size_t> dims = *tspace->get_dimensions();
		// Invoke kernel
		dft_kernel<T><<<dimGrid, dimBlock>>>(kspace->get_data_ptr()+i*elements,tspace->get_data_ptr()+i*t_elements,thrust::raw_pointer_cast(frequencies->data()),dims[0],dims[1], frequencies->size(),dte,dtt);
		hipDeviceSynchronize();
	CHECK_FOR_CUDA_ERROR();

	}

}

template<class T>
void Gadgetron::CSI_dftH(cuNDArray<complext<T> >* kspace,
		cuNDArray<complext<T> >* tspace, thrust::device_vector<T>* frequencies, T dtt, T dte) {
	size_t k_elements = kspace->get_size(0)*kspace->get_size(1);
	size_t elements = tspace->get_size(0)*tspace->get_size(1);

	size_t batches = tspace->get_number_of_elements()/elements;
	for (int i =0; i< batches; i++){
		int threadsPerBlock = std::min<int>(elements,cudaDeviceManager::Instance()->max_blockdim());
		dim3 dimBlock(threadsPerBlock);
		int totalBlocksPerGrid = (elements+threadsPerBlock-1)/threadsPerBlock;
		dim3 dimGrid(totalBlocksPerGrid);

		if (totalBlocksPerGrid > cudaDeviceManager::Instance()->max_griddim())
			throw std::runtime_error("CSIOperator: Input dimensions too large");

		//size_t batchSize = dimGrid.x*dimBlock.x;
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(dftH_kernel<T>),hipFuncCachePreferL1);

		std::vector<size_t> dims = *tspace->get_dimensions();

		// Invoke kernel
		dftH_kernel<T><<<dimGrid, dimBlock>>>(kspace->get_data_ptr()+i*k_elements,tspace->get_data_ptr()+i*elements,thrust::raw_pointer_cast(frequencies->data()),dims[0],dims[1], frequencies->size(),dte,dtt);
		CHECK_FOR_CUDA_ERROR();
	}
}

template EXPORTHYPER void Gadgetron::CSI_dft<float>(cuNDArray<float_complext>* kspace,cuNDArray<float_complext>* tspace, thrust::device_vector<float>* frequencies, float dtt, float dte);
template EXPORTHYPER void Gadgetron::CSI_dftH<float>(cuNDArray<float_complext>* kspace,cuNDArray<float_complext>* tspace, thrust::device_vector<float>* frequencies, float dtt, float dte);

