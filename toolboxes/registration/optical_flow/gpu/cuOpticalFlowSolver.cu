#include "hip/hip_runtime.h"
#include "cuOpticalFlowSolver.h"
#include "vector_td_utilities.h"
#include "check_CUDA.h"

#include <stdexcept>

namespace Gadgetron{

  //
  // Kernel prototype declarations
  //

  template<class REAL, unsigned int D> __global__ 
  void spatial_grad_kernel(const REAL*, const REAL*,REAL*,typename uint64d<D>::Type,unsigned int,unsigned int);

  template<class REAL, unsigned int D> __global__ 
  void temporal_grad_kernel(const REAL*, const REAL*,REAL*,typename uint64d<D>::Type,unsigned int,unsigned int);

  // There is some issue about Cuda defining min/max incompatibly...
  //

  template <class T> __host__ __device__ const T& _cuOF_max (const T& a, const T& b) {
    return (a<b)?b:a;
  }

  template <class T> __host__ __device__ const T& _cuOF_min (const T& a, const T& b) {
    return (a>b)?b:a;
  }

  template<class T, unsigned int D> void
  cuOpticalFlowSolver<T,D>::setup_grid( dim3 *blockDim, dim3* gridDim, 
					   unsigned int number_of_elements, 
					   unsigned int num_batches, 
					   bool use_2d_blocks, 
					   unsigned int num_unknowns )
  {
    int device;
    hipDeviceProp_t deviceProp; 
  
    if( hipGetDevice( &device ) != hipSuccess) {
      throw std::runtime_error("cuOpticalFlowSolver::setup_grid(): unable to determine current device");
    }
    
    if( hipGetDeviceProperties( &deviceProp, device ) != hipSuccess) {
      throw std::runtime_error("cuOpticalFlowSolver::setup_grid(): unable to query current device");
    }
    
    int max_blockdim = deviceProp.maxThreadsDim[0];
    int max_griddim  = deviceProp.maxGridSize[0];
    int warp_size    = deviceProp.warpSize;
    
    // For small arrays we keep the block dimension fairly small
    if( use_2d_blocks )
      *blockDim = dim3(((256/num_unknowns)/warp_size)*warp_size, num_unknowns);
    else
      *blockDim = dim3(256);
  
    *gridDim = dim3((number_of_elements+(blockDim->x*blockDim->y)-1)/(blockDim->x*blockDim->y), num_batches);

    // Extend block/grid dimensions for large arrays
    if( gridDim->x > max_griddim ){
      if( use_2d_blocks )
        blockDim->x = ((max_blockdim/num_unknowns)/warp_size)*warp_size;
      else
        blockDim->x = max_blockdim;
    
      gridDim->x = (number_of_elements+(blockDim->x*blockDim->y)-1)/(blockDim->x*blockDim->y);
    }

    if( gridDim->x > max_griddim ){
      gridDim->x = ((unsigned int)std::sqrt((T)number_of_elements)+(blockDim->x*blockDim->y)-1)/(blockDim->x*blockDim->y);
      gridDim->y *= ((number_of_elements+(blockDim->x*blockDim->y)*gridDim->x-1)/((blockDim->x*blockDim->y)*gridDim->x));
    }
   
    if( gridDim->x > max_griddim || gridDim->y > max_griddim ){      
      throw std::runtime_error("cuOpticalFlowSolver::setup_grid(): maximum grid dimensions exceeded");
    }
  }
  
  template<class T, unsigned int D> void
  cuOpticalFlowSolver<T,D>::core_grad_spatial( T *fixed_image, T *moving_image, T *gradient_image, 
						  typename uint64d<D>::Type matrix_size_moving, 
						  size_t number_of_batches_fixed, 
						  size_t number_of_batches_moving )
  {        
    unsigned int number_of_elements = prod(matrix_size_moving);
    dim3 blockDim; dim3 gridDim;

    setup_grid( &blockDim, &gridDim, number_of_elements, _cuOF_max(number_of_batches_moving, number_of_batches_fixed)*D );
    
    // Invoke kernel (spatial partial derivatives)
    spatial_grad_kernel<T,D><<< gridDim, blockDim >>>
      ( fixed_image, moving_image, gradient_image, matrix_size_moving, number_of_batches_fixed, number_of_batches_moving );
    
    CHECK_FOR_CUDA_ERROR();
  }
  
  template<class T, unsigned int D> void
  cuOpticalFlowSolver<T,D>::core_grad_temporal( T *fixed_image, T *moving_image, T *gradient_image, 
						   typename uint64d<D>::Type matrix_size_moving, 
						   size_t number_of_batches_fixed, 
						   size_t number_of_batches_moving )
  {        
    unsigned int number_of_elements = prod(matrix_size_moving);
    dim3 blockDim; dim3 gridDim;
    
    setup_grid( &blockDim, &gridDim, number_of_elements, _cuOF_max(number_of_batches_moving, number_of_batches_fixed) );
    
    // Invoke kernel (temporal partial derivative)
    temporal_grad_kernel<T,D><<< gridDim, blockDim >>>
      ( fixed_image, moving_image, gradient_image,
        matrix_size_moving, number_of_batches_fixed, number_of_batches_moving );
    
    CHECK_FOR_CUDA_ERROR();
  }
  
  // Helpers
  //

  template<unsigned int D> __device__ 
  typename uint64d<D>::Type compute_stride( unsigned int dim )
  {
    typename uint64d<D>::Type res;
  
    for( unsigned int d=0; d<D; d++ ){
      res.vec[d] = (d==dim) ? 1 : 0;
    }
    return res;
  }

  template<unsigned int D> __device__ 
  bool is_border_pixel_in_stride_dim_before( unsigned int dim, typename uint64d<D>::Type co, typename uint64d<D>::Type dims )
  {
    if( co.vec[dim] == 0 )
      return true;
    else
      return false;
  }

  template<unsigned int D> __device__ 
  bool is_border_pixel_in_stride_dim_after( unsigned int dim, typename uint64d<D>::Type co, typename uint64d<D>::Type dims )
  {
    if( co.vec[dim] == (dims.vec[dim]-1) )
      return true;
    else
      return false;
  }

  // Spatial partial derivatives
  //

  template<class REAL, unsigned int D> __global__ void
  spatial_grad_kernel( const REAL * __restrict__ fixed_image, const REAL * __restrict__ moving_image, REAL * __restrict__ gradient_image,
                       typename uint64d<D>::Type matrix_size, 
                       unsigned int num_batches_fixed, unsigned int num_batches_moving )
  {
    const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

    // Number of elements per partial derivate
    const unsigned int num_elements_per_batch = prod(matrix_size);
    const unsigned int num_elements_per_pdev_fixed = num_elements_per_batch*num_batches_fixed;
    const unsigned int num_elements_per_pdev_moving = num_elements_per_batch*num_batches_moving;

    // Total number of elements for all partial derivatives
    const unsigned int num_elements_total = _cuOF_max(num_elements_per_pdev_fixed, num_elements_per_pdev_moving)*D;
  
    if( idx < num_elements_total ){
    
      // The (minimum) index in the slowest varying output dimension determines which partial derivative to compute 
      const unsigned int stride_dim_fixed = idx/(num_elements_per_pdev_fixed);
      const unsigned int stride_dim_moving = idx/(num_elements_per_pdev_moving);
      const unsigned int stride_dim = _cuOF_min(stride_dim_fixed, stride_dim_moving);

      // Local index to the partial derivative
      const unsigned int idx_in_pdev_fixed = idx-stride_dim_fixed*num_elements_per_pdev_fixed;
      const unsigned int idx_in_pdev_moving = idx-stride_dim_moving*num_elements_per_pdev_moving;

      // Batch idx (second slowest varying dimension)   
      const unsigned int batch_idx_fixed = idx_in_pdev_fixed/num_elements_per_batch;
      const unsigned int batch_idx_moving = idx_in_pdev_moving/num_elements_per_batch;

      // Local index to the batch (should be identical for the fixed/moving image)
      const size_t idx_in_batch = idx_in_pdev_moving-batch_idx_moving*num_elements_per_batch;

      // Local co to the image
      const typename uint64d<D>::Type co = idx_to_co<D>( idx_in_batch, matrix_size );
 
      REAL res;
      unsigned int count = 0;

      //
      // Find partial derivatives using central differences
      //
    
      typename uint64d<D>::Type stride = compute_stride<D>(stride_dim);
    
      const unsigned int base_idx_moving = batch_idx_moving*num_elements_per_batch;
      const unsigned int base_idx_fixed = batch_idx_fixed*num_elements_per_batch;

      unsigned int stride_base_idx, fixed_idx, moving_idx;
     
      // Neighbor "plus stride" side
      if( !is_border_pixel_in_stride_dim_after<D>( stride_dim, co, matrix_size )){
        stride_base_idx = co_to_idx<D>(co+stride, matrix_size);
        count++;
      }
      else{
        stride_base_idx = idx_in_batch;
      }
    
      fixed_idx = stride_base_idx+base_idx_fixed;
      moving_idx = stride_base_idx+base_idx_moving;
    
      res = (fixed_image[fixed_idx]+moving_image[moving_idx])*REAL(0.5);

      // Neighbor "minus stride" side
      if( !is_border_pixel_in_stride_dim_before<D>( stride_dim, co, matrix_size )){
        stride_base_idx = co_to_idx<D>(co-stride, matrix_size);
        count++;
      }
      else{
        stride_base_idx = co_to_idx<D>(co, matrix_size);
      }
    
      fixed_idx = stride_base_idx+base_idx_fixed;
      moving_idx = stride_base_idx+base_idx_moving;
    
      res -= (fixed_image[fixed_idx]+moving_image[moving_idx])*REAL(0.5);

      if( count == 2 ) // Both neighbors exist
        res /= REAL(2);

      // Output result
      //
    
      gradient_image[idx] = res;
    }
  }

  // Temporal partial derivatives
  //

  template<class REAL, unsigned int D> __global__ void
  temporal_grad_kernel( const REAL * __restrict__ fixed_image, const REAL * __restrict__ moving_image, REAL * __restrict__ gradient_image,
                        typename uint64d<D>::Type matrix_size, 
                        unsigned int num_batches_fixed, unsigned int num_batches_moving )
  { 
    const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

    // Number of elements per partial derivate
    const unsigned int num_elements_per_batch = prod(matrix_size);
    const unsigned int num_elements_per_pdev_fixed = num_elements_per_batch*num_batches_fixed;
    const unsigned int num_elements_per_pdev_moving = num_elements_per_batch*num_batches_moving;

    // Total number of elements for all partial derivatives
    const unsigned int num_elements_total = _cuOF_max(num_elements_per_pdev_fixed, num_elements_per_pdev_moving);
  
    if( idx < num_elements_total ){
    
      const unsigned int stride_dim_fixed = idx/(num_elements_per_pdev_fixed);
      const unsigned int stride_dim_moving = idx/(num_elements_per_pdev_moving);

      // Local index to the partial derivative
      const unsigned int idx_in_pdev_fixed = idx-stride_dim_fixed*num_elements_per_pdev_fixed;
      const unsigned int idx_in_pdev_moving = idx-stride_dim_moving*num_elements_per_pdev_moving;

      // Batch idx (second slowest varying dimension)   
      const unsigned int batch_idx_fixed = idx_in_pdev_fixed/num_elements_per_batch;
      const unsigned int batch_idx_moving = idx_in_pdev_moving/num_elements_per_batch;

      // Local index to the batch (should be identical for the fixed/moving image)
      const unsigned int idx_in_batch = idx_in_pdev_moving-batch_idx_moving*num_elements_per_batch;

      const unsigned int base_idx_fixed = batch_idx_fixed*num_elements_per_batch;
      const unsigned int base_idx_moving = batch_idx_moving*num_elements_per_batch;
    
      // Ctr pixel
      const unsigned int fixed_idx = idx_in_batch+base_idx_fixed;
      const unsigned int moving_idx = idx_in_batch+base_idx_moving;
    
      const REAL res = moving_image[moving_idx]-fixed_image[fixed_idx];
    
      // Output result
      //
    
      gradient_image[idx] = res;        
    }    
  }

  // 
  // Template instantiation
  //

  template class EXPORTGPUREG cuOpticalFlowSolver<float,1>;
  template class EXPORTGPUREG cuOpticalFlowSolver<float,2>;
  template class EXPORTGPUREG cuOpticalFlowSolver<float,3>;
  template class EXPORTGPUREG cuOpticalFlowSolver<float,4>;

  template class EXPORTGPUREG cuOpticalFlowSolver<double,1>;
  template class EXPORTGPUREG cuOpticalFlowSolver<double,2>;
  template class EXPORTGPUREG cuOpticalFlowSolver<double,3>;
  template class EXPORTGPUREG cuOpticalFlowSolver<double,4>;
}
