#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2010, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.

When using this code in a scientific project, please cite one or all of the
following papers:
*  Daniel Ruijters and Philippe Th�venaz,
   GPU Prefilter for Accurate Cubic B-Spline Interpolation, 
   The Computer Journal, vol. 55, no. 1, pp. 15-20, January 2012.
   http://dannyruijters.nl/docs/cudaPrefilter3.pdf
*  Daniel Ruijters, Bart M. ter Haar Romeny, and Paul Suetens,
   Efficient GPU-Based Texture Interpolation using Uniform B-Splines,
   Journal of Graphics Tools, vol. 13, no. 4, pp. 61-69, 2008.
\*--------------------------------------------------------------------------*/

#ifndef _3D_CUBIC_BSPLINE_PREFILTER_H_
#define _3D_CUBIC_BSPLINE_PREFILTER_H_

#include <stdio.h>
#include "internal/cubicPrefilter_kernel.cu"

//--------------------------------------------------------------------------
// Global CUDA procedures
//--------------------------------------------------------------------------
template<class floatN>
__global__ static void SamplesToCoefficients3DX(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in x-direction
	const uint y = blockIdx.x * blockDim.x + threadIdx.x;
	const uint z = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = (z * height + y) * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr, width, sizeof(floatN));
}

template<class floatN>
__global__ static void SamplesToCoefficients3DY(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in y-direction
	const uint x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint z = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = z * height * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr + x, height, pitch);
}

template<class floatN>
__global__ static void SamplesToCoefficients3DZ(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in z-direction
	const uint x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = y * pitch;
	const uint slice = height * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr + x, depth, slice);
}

//--------------------------------------------------------------------------
// Exported functions
//--------------------------------------------------------------------------

//! Convert the voxel values into cubic b-spline coefficients
//! @param volume  pointer to the voxel volume in GPU (device) memory
//! @param pitch   width in bytes (including padding bytes)
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
template<class floatN>
extern void CubicBSplinePrefilter3D(floatN* volume, uint pitch, uint width, uint height, uint depth)
{
	// Try to determine the optimal block dimensions
	uint dimX = min(min(PowTwoDivider(width), PowTwoDivider(height)), 64);
	uint dimY = min(min(PowTwoDivider(depth), PowTwoDivider(height)), 512/dimX);
	dim3 dimBlock(dimX, dimY);

	// Replace the voxel values by the b-spline coefficients
	dim3 dimGridX(height / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DX<floatN><<<dimGridX, dimBlock>>>(volume, pitch, width, height, depth);
//	checkCudaErrors("SamplesToCoefficients3DX kernel failed");

	dim3 dimGridY(width / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DY<floatN><<<dimGridY, dimBlock>>>(volume, pitch, width, height, depth);
//	checkCudaErrors("SamplesToCoefficients3DY kernel failed");

	dim3 dimGridZ(width / dimBlock.x, height / dimBlock.y);
	SamplesToCoefficients3DZ<floatN><<<dimGridZ, dimBlock>>>(volume, pitch, width, height, depth);
//	checkCudaErrors("SamplesToCoefficients3DZ kernel failed");
}

//! Convert the voxel values into cubic b-spline coefficients
//! @param volume  pointer to the voxel volume in GPU (device) memory
//! @param pitch   width in bytes (including padding bytes)
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
//! @note Prints stopwatch feedback
template<class floatN>
extern void CubicBSplinePrefilter3DTimer(floatN* volume, uint pitch, uint width, uint height, uint depth)
{
	//printf("\nCubic B-Spline Prefilter timer:\n");

	// Try to determine the optimal block dimensions
	uint dimX = min(min(PowTwoDivider(width), PowTwoDivider(height)), 64);
	uint dimY = min(min(PowTwoDivider(depth), PowTwoDivider(height)), 512/dimX);
	dim3 dimBlock(dimX, dimY);

	// Replace the voxel values by the b-spline coefficients
	dim3 dimGridX(height / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DX<floatN><<<dimGridX, dimBlock>>>(volume, pitch, width, height, depth);

	dim3 dimGridY(width / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DY<floatN><<<dimGridY, dimBlock>>>(volume, pitch, width, height, depth);

	dim3 dimGridZ(width / dimBlock.x, height / dimBlock.y);
	SamplesToCoefficients3DZ<floatN><<<dimGridZ, dimBlock>>>(volume, pitch, width, height, depth);

}

#endif  //_3D_CUBIC_BSPLINE_PREFILTER_H_
