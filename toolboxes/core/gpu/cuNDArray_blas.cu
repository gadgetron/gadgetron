#include "hip/hip_runtime.h"
#include "cuNDArray_blas.h"
#include "complext.h"
#include "GadgetronCuException.h"
#include "cudaDeviceManager.h"

#include <hipblas.h>

namespace Gadgetron{

#define CUBLAS_CALL(fun) {hipblasStatus_t err = fun; if (err != HIPBLAS_STATUS_SUCCESS) {throw cuda_error(gadgetron_getCublasErrorString(err));}}

  //NRM2
  //

  template<class T> hipblasStatus_t cublas_axpy(hipblasHandle_t hndl, int n, const T* a , const T* x , int incx,  T* y, int incy);
  template<class T> hipblasStatus_t cublas_dot(hipblasHandle_t, int, const T*, int, const  T*, int, T*, bool cc = true);
  template<class T> hipblasStatus_t cublas_nrm2(hipblasHandle_t, int, const T*, int, typename realType<T>::Type *result);
  template<class T> hipblasStatus_t cublas_amax(hipblasHandle_t handle, int n,const T *x, int incx, int *result);
  template<class T> hipblasStatus_t cublas_amin(hipblasHandle_t handle, int n,const T *x, int incx, int *result);
  template<class T> hipblasStatus_t cublas_asum(hipblasHandle_t handle, int n,const T *x, int incx, typename realType<T>::Type *result);

  template<> hipblasStatus_t cublas_nrm2<float>(hipblasHandle_t hndl, int n, const float*  x, int inc, float* res){
    return hipblasSnrm2(hndl,n,x,inc,res);
  }

  template<> hipblasStatus_t cublas_nrm2<double>(hipblasHandle_t hndl, int n, const double*  x, int inc, double* res){
    return hipblasDnrm2(hndl,n,x,inc,res);
  }

  template<> hipblasStatus_t cublas_nrm2<float_complext>(hipblasHandle_t hndl, int n, const float_complext*  x, int inc, float* res){
    return hipblasScnrm2(hndl,n,(const hipComplex*)x,inc,res);
  }

  template<> hipblasStatus_t cublas_nrm2<double_complext>(hipblasHandle_t hndl, int n, const double_complext*  x, int inc, double* res){
    return hipblasDznrm2(hndl,n,(const hipDoubleComplex*) x,inc,res);
  }

  //DOT
  //

  template<> hipblasStatus_t cublas_dot<float>(hipblasHandle_t hndl, int n , const float* x , int incx, const  float* y , int incy, float* res, bool cc){
    return hipblasSdot( hndl, n, x, incx, y, incy, res);
  }

  template<> hipblasStatus_t cublas_dot<double>(hipblasHandle_t hndl, int n , const double* x , int incx, const  double* y , int incy, double* res, bool cc){
    return hipblasDdot( hndl, n, x, incx, y, incy, res);
  }

  template<> hipblasStatus_t cublas_dot<float_complext>(hipblasHandle_t hndl, int n , const float_complext* x ,
										int incx, const  float_complext* y , int incy, float_complext* res, bool cc){
    if(cc)
      return hipblasCdotc( hndl, n, (const hipComplex*) x, incx, (const hipComplex*) y, incy, (hipComplex*) res);
    else
      return hipblasCdotu( hndl, n, (const hipComplex*) x, incx, (const hipComplex*) y, incy, (hipComplex*) res);
  }

  template<> hipblasStatus_t cublas_dot<double_complext>(hipblasHandle_t hndl, int n , const double_complext* x ,
										 int incx, const  double_complext* y , int incy, double_complext* res, bool cc){
    if(cc)
      return hipblasZdotc( hndl, n, (const hipDoubleComplex*) x, incx, (const hipDoubleComplex*) y, incy, (hipDoubleComplex*) res);
    else
      return hipblasZdotu( hndl, n, (const hipDoubleComplex*) x, incx, (const hipDoubleComplex*) y, incy, (hipDoubleComplex*) res);
  }

  // AXPY
  //

  template<> hipblasStatus_t cublas_axpy<float>(hipblasHandle_t hndl , int n , const float* a , const float* x , int incx ,  float* y , int incy){
    return hipblasSaxpy(hndl,n,a,x,incx,y,incy);
  }

  template<> hipblasStatus_t cublas_axpy<double>(hipblasHandle_t hndl , int n , const double* a , const double* x , int incx ,  double* y , int incy){
    return hipblasDaxpy(hndl,n,a,x,incx,y,incy);
  }

  template<> hipblasStatus_t cublas_axpy<float_complext>(hipblasHandle_t hndl , int n , const float_complext* a , const float_complext* x , int incx ,  float_complext* y , int incy){
    return hipblasCaxpy(hndl,n,(const hipComplex*) a, (const hipComplex*) x,incx, (hipComplex*)y,incy);
  }

  template<> hipblasStatus_t cublas_axpy<double_complext>(hipblasHandle_t hndl , int n , const double_complext* a , const double_complext* x , int incx ,  double_complext* y , int incy){
    return hipblasZaxpy(hndl,n,(const hipDoubleComplex*) a, (const hipDoubleComplex*) x,incx, (hipDoubleComplex*)y,incy);
  }

  //SUM
  //

  template<> hipblasStatus_t cublas_asum<float>(hipblasHandle_t hndl, int n,const float *x, int incx, float *result){
    return hipblasSasum(hndl,n,x,incx,result);
  }

  template<> hipblasStatus_t cublas_asum<double>(hipblasHandle_t hndl, int n,const double *x, int incx, double *result){
    return hipblasDasum(hndl,n,x,incx,result);
  }

  template<> hipblasStatus_t cublas_asum<float_complext>(hipblasHandle_t hndl, int n,const float_complext *x, int incx, float *result){
    return hipblasScasum(hndl,n,(const hipComplex*) x,incx,result);
  }

  template<> hipblasStatus_t cublas_asum<double_complext>(hipblasHandle_t hndl, int n,const double_complext *x, int incx, double *result){
    return hipblasDzasum(hndl,n,(const hipDoubleComplex*) x,incx,result);
  }

  //AMIN
  //

  template<> hipblasStatus_t cublas_amin<float>(hipblasHandle_t hndl, int n,const float *x, int incx, int *result){
    return hipblasIsamin(hndl,n,x,incx,result);
  }

  template<> hipblasStatus_t cublas_amin<double>(hipblasHandle_t hndl, int n,const double *x, int incx, int *result){
    return hipblasIdamin(hndl,n,x,incx,result);
  }

  template<> hipblasStatus_t cublas_amin<float_complext>(hipblasHandle_t hndl, int n,const float_complext *x, int incx, int *result){
    return hipblasIcamin(hndl,n, (const hipComplex* ) x,incx,result);
  }

  template<> hipblasStatus_t cublas_amin<double_complext>(hipblasHandle_t hndl, int n,const double_complext *x, int incx, int *result){
    return hipblasIzamin(hndl,n, (const hipDoubleComplex* ) x,incx,result);
  }

  //AMAX
  //

  template<> hipblasStatus_t cublas_amax<float>(hipblasHandle_t hndl, int n,const float *x, int incx, int *result){
    return hipblasIsamax(hndl,n,x,incx,result);
  }

  template<> hipblasStatus_t cublas_amax<double>(hipblasHandle_t hndl, int n,const double *x, int incx, int *result){
    return hipblasIdamax(hndl,n,x,incx,result);
  }

  template<> hipblasStatus_t cublas_amax<float_complext>(hipblasHandle_t hndl, int n,const float_complext *x, int incx, int *result){
    return hipblasIcamax(hndl,n, (const hipComplex* ) x,incx,result);
  }

  template<> hipblasStatus_t cublas_amax<double_complext>(hipblasHandle_t hndl, int n,const double_complext *x, int incx, int *result){
    return hipblasIzamax(hndl,n, (const hipDoubleComplex* ) x,incx,result);
  }

  template <class T> typename realType<T>::Type nrm2(cuNDArray<T>* arr , size_t batchSize ) 
  {
    if( arr == 0x0 )
        throw std::runtime_error("Gadgetron::nrm2(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    typedef typename realType<T>::Type REAL;
    REAL ret = 0;

    // If number of elements in the array is greater than batchSize break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = arr->get_number_of_elements() / batchSize + 1;
    int remainder = arr->get_number_of_elements() - batchSize * (num_splits - 1);
    auto handle = cudaDeviceManager::Instance()->lockHandle(device);

    for (int ii = 0; ii < num_splits; ii++) {

        REAL val;

        CUBLAS_CALL(cublas_nrm2<T>(handle,
                                   (ii == num_splits - 1) ? remainder : batchSize, // n number of elements
                                   arr->get_data_ptr() + batchSize * ii, 1, &val));


        if (ii == 0)
            ret = val;
        else
            ret = sqrt(pow(ret, 2) + pow(val, 2));
    }

    cudaDeviceManager::Instance()->unlockHandle(device);

    return ret;
  }

  template <class T> T dot(cuNDArray<T>* arr1, cuNDArray<T>* arr2, size_t batchSize , bool cc) 
  {
    if (arr1 == 0x0 || arr2 == 0x0)
        throw std::runtime_error("Gadgetron::dot(): Invalid input array");

    if (arr1->get_number_of_elements() != arr2->get_number_of_elements())
        throw std::runtime_error("Gadgetron::dot(): Array sizes mismatch");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    T ret = 0;

    // If number of elements in the array is greater than batchSize break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = arr1->get_number_of_elements() / batchSize + 1;
    int remainder = arr1->get_number_of_elements() - batchSize * (num_splits - 1);
    auto handle = cudaDeviceManager::Instance()->lockHandle(device);

    for (int ii = 0; ii < num_splits; ii++) {

        T val;
        CUBLAS_CALL(cublas_dot(handle,
                               (ii == num_splits - 1) ? remainder : batchSize, // n number of elements
                               arr1->get_data_ptr() + batchSize * ii, 1, arr2->get_data_ptr() + batchSize * ii, 1, &val,
                               cc));

        if (ii == 0)
            ret = val;
        else
            ret += val;
    }
    
    cudaDeviceManager::Instance()->unlockHandle(device);

    return ret;
  }

  template <class T> void axpy(T a, cuNDArray<T>* x, cuNDArray<T>* y, size_t batchSize ) 
  {
    if (x == 0x0 || y == 0x0)
        throw std::runtime_error("Gadgetron::axpy(): Invalid input array");

    if (x->get_number_of_elements() != y->get_number_of_elements())
        throw std::runtime_error("Gadgetron::axpy(): Array sizes mismatch");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();

    // If number of elements in the array is greater than batchSize break it up and perform calculations this is done to
    // support large data arrays
    int num_splits = x->get_number_of_elements() / batchSize + 1;
    int remainder = x->get_number_of_elements() - batchSize * (num_splits - 1);
    auto handle = cudaDeviceManager::Instance()->lockHandle(device);

    for (int ii = 0; ii < num_splits; ii++) {

        CUBLAS_CALL(cublas_axpy(handle,
                                (ii == num_splits - 1) ? remainder : batchSize, &a, x->get_data_ptr() + batchSize * ii, 1,
                                y->get_data_ptr() + batchSize * ii, 1));

    }
    cudaDeviceManager::Instance()->unlockHandle(device);

  }

  template <class T> void axpy(T a, cuNDArray<complext<T>>* x, cuNDArray<complext<T>>* y, size_t batchSize ) { axpy(complext<T>(a), x, y, batchSize); }

  template <class T> typename realType<T>::Type asum(cuNDArray<T>* x, size_t batchSize ) 
  {
    if (x == 0x0)
        throw std::runtime_error("Gadgetron::asum(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    typename realType<T>::Type result = 0;
    
    // If number of elements in the array is greater than batchSize break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = x->get_number_of_elements() / batchSize + 1;
    int remainder = x->get_number_of_elements() - batchSize * (num_splits - 1);
    auto handle = cudaDeviceManager::Instance()->lockHandle(device);

    for (int ii = 0; ii < num_splits; ii++) {
        typename realType<T>::Type interim_result;

        CUBLAS_CALL(cublas_asum(handle,
                                (ii == num_splits - 1) ? remainder : batchSize, // n number of elements
                                x->get_data_ptr() + batchSize * ii, 1, &interim_result));


        if (ii == 0)
            result = interim_result;
        else
            result += interim_result;
    }
    cudaDeviceManager::Instance()->unlockHandle(device);

    return result;
  }

  template <class T> size_t amin(cuNDArray<T>* x, size_t batchSize ) 
  {
    if (x == 0x0)
        throw std::runtime_error("Gadgetron::amin(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    size_t result = x->get_number_of_elements() + 1;

    // If number of elements in the array is greater than batchSize break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = x->get_number_of_elements() / batchSize + 1;
    int remainder = x->get_number_of_elements() - batchSize * (num_splits - 1);
    auto handle = cudaDeviceManager::Instance()->lockHandle(device);
    T saved_value = T(0);
    
    // Since operators cannot be overloaded inside functions - this lambda function should do the comparison and mimic cublas comparisons
    auto lessThan = [](auto c1, auto c2)
    {
        if constexpr (std::is_same_v<decltype(c1),float> || std::is_same_v<decltype(c1),double>)
            return (abs(c1) < abs(c2));
        else if constexpr (std::is_same_v<decltype(c1),complext<float>> || std::is_same_v<decltype(c1),complext<double>>)
            return (abs(c1.real()) + abs(c1.imag())) < (abs(c2.real()) + abs(c2.imag()));
    };

    for (int ii = 0; ii < num_splits; ii++) {

        int interim_result;

        CUBLAS_CALL(cublas_amin(handle,
                                (ii == num_splits - 1) ? remainder : batchSize, // n number of elements
                                x->get_data_ptr() + batchSize * ii, 1, &interim_result));

        auto interim_value = (*x)[batchSize * ii + (size_t)interim_result - 1];

        if (ii == 0)
            result = (size_t)interim_result - 1;
        else if (lessThan((interim_value), (saved_value)))
            result = batchSize * ii + (size_t)interim_result - 1;
        
        saved_value = (*x)[result];

    }
    cudaDeviceManager::Instance()->unlockHandle(device);
    if (result > x->get_number_of_elements()) {
        throw std::runtime_error("Gadgetron::amin(): computed index is out of bounds");
    }

    return result; // result - 1;
  }

  template <class T> size_t amax(cuNDArray<T>* x, size_t batchSize ) 
  {
    if (x == 0x0)
        throw std::runtime_error("Gadgetron::amax(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    size_t result = x->get_number_of_elements() + 1;

    // If number of elements in the array is greater than batchSize break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = x->get_number_of_elements() / batchSize + 1;
    int remainder = x->get_number_of_elements() - batchSize * (num_splits - 1);
    auto handle = cudaDeviceManager::Instance()->lockHandle(device);
    T saved_value = T(0);

    // Since operators cannot be overloaded inside functions - this lambda function should do the comparison and mimic cublas comparisons
    auto greaterThan = [](auto c1, auto c2)
    {
        if constexpr (std::is_same_v<decltype(c1),float> || std::is_same_v<decltype(c1),double>)
            return (abs(c1) > abs(c2));
        else if constexpr (std::is_same_v<decltype(c1),complext<float>> || std::is_same_v<decltype(c1),complext<double>>)
            return (abs(c1.real()) + abs(c1.imag())) > (abs(c2.real()) + abs(c2.imag()));
    };

    for (int ii = 0; ii < num_splits; ii++) {

        int interim_result;
        CUBLAS_CALL(cublas_amax(
            handle,
            (ii == num_splits - 1) ? remainder : batchSize, // n number of elements (int)x->get_number_of_elements(),
            x->get_data_ptr() + batchSize * ii, 1, &interim_result));
       
         auto interim_value = (*x)[batchSize * ii + (size_t)interim_result - 1];

        if (ii == 0)
            result = (size_t)interim_result - 1;
        else if (greaterThan((interim_value), (saved_value)))
            result = batchSize * ii + (size_t)interim_result - 1;
        
        saved_value = (*x)[result];

    }
            cudaDeviceManager::Instance()->unlockHandle(device);
    if (result > x->get_number_of_elements()) {
        throw std::runtime_error("Gadgetron::amax(): computed index is out of bounds");
    }

    return result; //(size_t)result - 1;
  }

  std::string gadgetron_getCublasErrorString(hipblasStatus_t err) 
  {
    switch (err) {
    case HIPBLAS_STATUS_NOT_INITIALIZED:
        return "NOT INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
        return "ALLOC FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
        return "INVALID VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
        return "ARCH MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
        return "MAPPING ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
        return "EXECUTION FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
        return "INTERNAL ERROR";
    case HIPBLAS_STATUS_SUCCESS:
        return "SUCCES";
    default:
        return "UNKNOWN CUBLAS ERROR";
    }
  }

  //
  // Instantiation
  //

  template float dot(cuNDArray<float>*, cuNDArray<float>*, size_t, bool);
  template float nrm2(cuNDArray<float>*, size_t);
  template void axpy(float, cuNDArray<float>*, cuNDArray<float>*, size_t);
  template size_t amin(cuNDArray<float>*, size_t);
  template size_t amax(cuNDArray<float>*, size_t);
  template float asum(cuNDArray<float>*, size_t);

  template double dot(cuNDArray<double>*, cuNDArray<double>*, size_t, bool);
  template double nrm2(cuNDArray<double>*, size_t);
  template void axpy(double, cuNDArray<double>*, cuNDArray<double>*, size_t);
  template size_t amin(cuNDArray<double>*, size_t);
  template size_t amax(cuNDArray<double>*, size_t);
  template double asum(cuNDArray<double>*, size_t);

  template float_complext dot(cuNDArray<float_complext>*, cuNDArray<float_complext>*, size_t, bool);
  template float nrm2(cuNDArray<float_complext>*, size_t);
  template void axpy(float_complext, cuNDArray<float_complext>*, cuNDArray<float_complext>*, size_t);
  template void axpy(float, cuNDArray<float_complext>*, cuNDArray<float_complext>*, size_t);
  template size_t amin(cuNDArray<float_complext>*, size_t);
  template size_t amax(cuNDArray<float_complext>*, size_t);
  template float asum(cuNDArray<float_complext>*, size_t);

  template double_complext dot(cuNDArray<double_complext>*, cuNDArray<double_complext>*, size_t, bool);
  template double nrm2(cuNDArray<double_complext>*, size_t);
  template void axpy(double_complext, cuNDArray<double_complext>*, cuNDArray<double_complext>*, size_t);
  template void axpy(double, cuNDArray<double_complext>*, cuNDArray<double_complext>*, size_t);
  template size_t amin(cuNDArray<double_complext>*, size_t);
  template size_t amax(cuNDArray<double_complext>*, size_t);
  template double asum(cuNDArray<double_complext>*, size_t);
} // namespace Gadgetron
