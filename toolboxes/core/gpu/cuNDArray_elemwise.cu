#include "hip/hip_runtime.h"
#include "cuNDArray_elemwise.h"
#include "cuNDArray_operators.h"
#include "cuNDArray_blas.h"
#include "complext.h"

#include <complex>
#include <thrust/functional.h>

using namespace Gadgetron;
//using namespace std;

template<typename T> struct cuNDA_abs : public thrust::unary_function<T,typename realType<T>::Type>
{
  __device__ typename Gadgetron::realType<T>::Type operator()(const T &x) const {return abs(x);}
};

template<class T> boost::shared_ptr< cuNDArray<typename realType<T>::Type> > 
Gadgetron::abs( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::abs(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<typename realType<T>::Type> > result(new cuNDArray<typename realType<T>::Type>());
  result->create(x->get_dimensions());
  thrust::device_ptr<typename realType<T>::Type> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_abs<T>());
  return result;
}

template<class T> void 
Gadgetron::abs_inplace( cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::abs_inplace(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_abs<T>());
}  
  
template<typename T> struct cuNDA_abs_square : public thrust::unary_function<T,typename realType<T>::Type>
{
  __device__ typename Gadgetron::realType<T>::Type operator()(const T &x) const 
  { 
    typename realType<T>::Type tmp = abs(x);
    return tmp*tmp;
  }
};

template<class T> boost::shared_ptr< cuNDArray<typename realType<T>::Type> > 
Gadgetron::abs_square( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::abs_square(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<typename realType<T>::Type> > result(new cuNDArray<typename realType<T>::Type>());
  result->create(x->get_dimensions());
  thrust::device_ptr<typename realType<T>::Type> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_abs_square<T>());
  return result;
}

template<typename T> struct cuNDA_sqrt : public thrust::unary_function<T,T>
{
  __device__ T operator()(const T &x) const {return sqrt(x);}
};

template<class T> boost::shared_ptr< cuNDArray<T> > 
Gadgetron::sqrt( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::sqrt(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<T> > result(new cuNDArray<T>());
  result->create(x->get_dimensions());
  thrust::device_ptr<T> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_sqrt<T>());
  return result;
}

template<class T> void 
Gadgetron::sqrt_inplace( cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::sqrt_inplace(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_sqrt<T>());
}
 
template<typename T> struct cuNDA_square : public thrust::unary_function<T,T>
{
  __device__ T operator()(const T &x) const {return x*x;}
};

template<class T> boost::shared_ptr< cuNDArray<T> > Gadgetron::square( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::square(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<T> > result(new cuNDArray<T>());
  result->create(x->get_dimensions());
  thrust::device_ptr<T> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_square<T>());
  return result;
}

template<class T> void 
Gadgetron::square_inplace( cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::square_inplace(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_square<T>());
}  

template<typename T> struct cuNDA_reciprocal : public thrust::unary_function<T,T>
{
  __device__ T operator()(const T &x) const {return T(1)/x;}
};

template<class T> boost::shared_ptr< cuNDArray<T> > Gadgetron::reciprocal( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::reciprocal(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<T> > result(new cuNDArray<T>());
  result->create(x->get_dimensions());
  thrust::device_ptr<T> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_reciprocal<T>());
  return result;
}

template<class T> void 
Gadgetron::reciprocal_inplace( cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::reciprocal_inplace(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_reciprocal<T>());
}  
 
template<typename T> struct cuNDA_reciprocal_sqrt : public thrust::unary_function<T,T>
{
  __device__ T operator()(const T &x) const {return T(1)/sqrt(x);}
};

template<class T> boost::shared_ptr< cuNDArray<T> > Gadgetron::reciprocal_sqrt( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::reciprocal_sqrt(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<T> > result(new cuNDArray<T>());
  result->create(x->get_dimensions());
  thrust::device_ptr<T> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_reciprocal_sqrt<T>());
  return result;
}

template<class T> void 
Gadgetron::reciprocal_sqrt_inplace( cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::reciprocal_sqrt_inplace(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_reciprocal_sqrt<T>());
}  

template<typename T> struct cuNDA_sgn : public thrust::unary_function<T,T>
{
  __device__ T operator()(const T &x) const {return sgn(x);}
};

template<class T> boost::shared_ptr< cuNDArray<T> > Gadgetron::sgn( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::sgn(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<T> > result(new cuNDArray<T>());
  result->create(x->get_dimensions());
  thrust::device_ptr<T> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_sgn<T>());
  return result;
}

template<class T> void 
Gadgetron::sgn_inplace( cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::sgn_inplace(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_sgn<T>());
}  
 
template<typename T> struct cuNDA_real : public thrust::unary_function<T,typename realType<T>::Type>
{
  __device__ typename realType<T>::Type operator()(const T &x) const {return real(x);}
};

template<class T> boost::shared_ptr< cuNDArray<typename realType<T>::Type> > 
Gadgetron::real( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::real(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<typename realType<T>::Type> > result(new cuNDArray<typename realType<T>::Type>());
  result->create(x->get_dimensions());
  thrust::device_ptr<typename realType<T>::Type> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_real<T>());
  return result;
}

template <typename T> struct cuNDA_imag : public thrust::unary_function<T,typename realType<T>::Type>
{
  __device__ typename realType<T>::Type operator()(const T &x) const {return imag(x);}
};

template<class T> boost::shared_ptr< cuNDArray<typename realType<T>::Type> > 
Gadgetron::imag( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::imag(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<typename realType<T>::Type> > result(new cuNDArray<typename realType<T>::Type>());
  result->create(x->get_dimensions());
  thrust::device_ptr<typename realType<T>::Type> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_imag<T>());
  return result;
}

template <typename T> struct cuNDA_conj : public thrust::unary_function<T,T>
{
  __device__ T operator()(const T &x) const {return conj(x);}
};

template<class T> boost::shared_ptr< cuNDArray<T> > 
Gadgetron::conj( const cuNDArray<T> *x )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::conj(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<T> > result(new cuNDArray<T>());
  result->create(x->get_dimensions());
  thrust::device_ptr<T> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_conj<T>());
  return result;
}

template <typename T> struct cuNDA_real_to_complex : public thrust::unary_function<typename realType<T>::Type,T>
{
  __device__ T operator()(const typename realType<T>::Type &x) const {return T(x);}
};

template<class T> boost::shared_ptr< cuNDArray<T> > 
Gadgetron::real_to_complex( const cuNDArray<typename realType<T>::Type> *x )
{
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::real_to_complex(): Invalid input array");
   
  boost::shared_ptr< cuNDArray<T> > result(new cuNDArray<T>());
  result->create(x->get_dimensions());
  thrust::device_ptr<T> resPtr = result->get_device_ptr();
  thrust::device_ptr<typename realType<T>::Type> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_real_to_complex<T>());
  return result;
}

template <typename T,typename T2> struct cuNDA_convert_to : public thrust::unary_function<T,T2>
{
  __device__ T2 operator()(T &x) const {return T2(x);}
};

template <typename T,typename T2> struct cuNDA_convert_to<complext<T>,complext<T2> > : public thrust::unary_function<complext<T>,complext<T2> >
{
  __device__ complext<T2> operator()(complext<T> &x) const {return complext<T2>(x._real,x._imag);}
};

template<class T, class T2> boost::shared_ptr< cuNDArray<T2> >
Gadgetron::convert_to( const cuNDArray<T> *x )
{
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::convert_to(): Invalid input array");

  boost::shared_ptr< cuNDArray<T2> > result(new cuNDArray<T2>());
  result->create(x->get_dimensions());
  thrust::device_ptr<T2> resPtr = result->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_convert_to<T,T2>());
  return result;
}

template<class T, class T2> void
Gadgetron::convert_to( cuNDArray<T> *x ,cuNDArray<T2> * y)
{
  if( x == 0x0 || !x->dimensions_equal(y))
    throw std::runtime_error("Gadgetron::convert_to(): Invalid input array");
  thrust::device_ptr<T2> resPtr = y->get_device_ptr();
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_convert_to<T,T2>());
}

template<class T> void Gadgetron::clear( cuNDArray<T> *x )
{
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::clear(): Invalid input array");

  if ( x->get_number_of_elements() > 0 )
  {
    hipMemset(x->get_data_ptr(),0,sizeof(T)*x->get_number_of_elements());
  }
}

template<class T> void 
Gadgetron::fill( cuNDArray<T> *x, T val )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::fill(): Invalid input array");
  
  thrust::device_ptr<T> devPtr = x->get_device_ptr();
  thrust::fill(devPtr,devPtr+x->get_number_of_elements(),val);
}  

template<typename T> struct cuNDA_clamp : public thrust::unary_function<T,T>
{
  cuNDA_clamp( T _min, T _max, T _min_val, T _max_val ) : min(_min), max(_max),min_val(_min_val), max_val(_max_val) {}
  __device__ T operator()(const T &x) const 
  {
    if( x < min ) return min_val;
    else if ( x >= max) return max_val;
    else return x;
  }
  T min, max;
  T min_val, max_val;
};

template<typename T> struct cuNDA_clamp< complext<T> > : public thrust::unary_function< complext<T>, complext<T> >
{
	cuNDA_clamp( T _min, T _max, complext<T> _min_val, complext<T> _max_val ) : min(_min), max(_max),min_val(_min_val), max_val(_max_val) {}
  __device__ complext<T> operator()(const complext<T> &x) const 
  {
    if( real(x) < min ) return min_val;
    else if ( real(x) >= max) return max_val;
    else return complext<T>(real(x));
  }
  T min, max;
  complext<T> min_val, max_val;
};

template<class T> void 
Gadgetron::clamp( cuNDArray<T> *x, typename realType<T>::Type min, typename realType<T>::Type max, T min_val, T max_val)
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::clamp(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_clamp<T>(min, max,min_val, max_val));
}  

template<class T> void 
Gadgetron::clamp( cuNDArray<T> *x, typename realType<T>::Type min, typename realType<T>::Type max)
{
    clamp(x,min,max,T(min),T(max));
}

template<typename T> struct cuNDA_clamp_min : public thrust::unary_function<T,T>
{
  cuNDA_clamp_min( T _min ) : min(_min) {}
  __device__ T operator()(const T &x) const 
  {
    if( x < min ) return min;
    else return x;
  }
  T min;
};

template<typename T> struct cuNDA_clamp_min< complext<T> > : public thrust::unary_function< complext<T>, complext<T> >
{
  cuNDA_clamp_min( T _min ) : min(_min) {}
  __device__ complext<T> operator()(const complext<T> &x) const 
  {
    if( real(x) < min ) return complext<T>(min);
    else return complext<T>(real(x));
  }
  T min;
};

template<class T> void 
Gadgetron::clamp_min( cuNDArray<T> *x, typename realType<T>::Type min )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::clamp_min(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_clamp_min<T>(min));
}  

template<typename T> struct cuNDA_clamp_max : public thrust::unary_function<T,T>
{
  cuNDA_clamp_max( T _max ) : max(_max) {}
  __device__ T operator()(const T &x) const 
  {
    if( x > max ) return max;
    else return x;
  }
  T max;
};

template<typename T> struct cuNDA_clamp_max< complext<T> > : public thrust::unary_function< complext<T>, complext<T> >
{
  cuNDA_clamp_max( T _max ) : max(_max) {}
  __device__ complext<T> operator()(const complext<T> &x) const 
  {
    if( real(x) > max ) return complext<T>(max);
    else return complext<T>(real(x));
  }
  T max;
};

template<class T> void 
Gadgetron::clamp_max( cuNDArray<T> *x, typename realType<T>::Type max )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::clamp_max(): Invalid input array");
   
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),xPtr,cuNDA_clamp_max<T>(max));
}  

template<class T> void 
Gadgetron::normalize( cuNDArray<T> *x, typename realType<T>::Type val )
{
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::normalize(): Invalid input array");
  
  size_t max_idx = amax(x);
  T max_val_before;
  CUDA_CALL(hipMemcpy(&max_val_before, &x->get_data_ptr()[max_idx], sizeof(T), hipMemcpyDeviceToHost));
  typename realType<T>::Type scale = val/abs(max_val_before);
  *x *= scale;
}


template<typename T> struct cuNDA_shrink1 : public thrust::unary_function<T,T>
{
  cuNDA_shrink1( typename realType<T>::Type _gamma ) : gamma(_gamma) {}
  __device__ T operator()(const T &x) const {
    typename realType<T>::Type absX = abs(x);
    T sgnX = (absX <= typename realType<T>::Type(0)) ? T(0) : x/absX;
    return sgnX*max(absX-gamma, typename realType<T>::Type(0));
  }
  typename realType<T>::Type gamma;
};

template<class T> void 
Gadgetron::shrink1( cuNDArray<T> *x, typename realType<T>::Type gamma, cuNDArray<T> *out )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::shrink1(): Invalid input array");
  
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::device_ptr<T> outPtr = (out == 0x0) ? x->get_device_ptr() : out->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),outPtr,cuNDA_shrink1<T>(gamma));
}

template<typename T> struct cuNDA_pshrink : public thrust::unary_function<T,T>
{
  cuNDA_pshrink( typename realType<T>::Type _gamma, typename realType<T>::Type _p ) : gamma(_gamma),p(_p) {}
  __device__ T operator()(const T &x) const {
    typename realType<T>::Type absX = abs(x);
    T sgnX = (absX <= typename realType<T>::Type(0)) ? T(0) : x/absX;
    return sgnX*max(absX-gamma*pow(absX,p-1), typename realType<T>::Type(0));
  }
  typename realType<T>::Type gamma;
  typename realType<T>::Type p;
};

template<class T> void
Gadgetron::pshrink( cuNDArray<T> *x, typename realType<T>::Type gamma,typename realType<T>::Type p, cuNDArray<T> *out )
{
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::shrink1(): Invalid input array");

  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::device_ptr<T> outPtr = (out == 0x0) ? x->get_device_ptr() : out->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),outPtr,cuNDA_pshrink<T>(gamma,p));
}  

template<typename T> struct cuNDA_shrinkd : public thrust::binary_function<T,typename realType<T>::Type,T>
{
  cuNDA_shrinkd( typename realType<T>::Type _gamma ) : gamma(_gamma) {}
  __device__ T operator()(const T &x, const typename realType<T>::Type &s) const {
  	T xs = (s <= typename realType<T>::Type(0)) ? T(0) : x/s;
    return xs*max(s-gamma,typename realType<T>::Type(0));
  }
  typename realType<T>::Type gamma;
};

template<class T> void 
Gadgetron::shrinkd( cuNDArray<T> *x, cuNDArray<typename realType<T>::Type> *s, typename realType<T>::Type gamma, cuNDArray<T> *out )
{ 
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::shrinkd(): Invalid input array");
  
  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::device_ptr<T> outPtr = (out == 0x0) ? x->get_device_ptr() : out->get_device_ptr();
  thrust::device_ptr<typename realType<T>::Type> sPtr = s->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),sPtr,outPtr,cuNDA_shrinkd<T>(gamma));
}  


template<typename T> struct cuNDA_pshrinkd : public thrust::binary_function<T,typename realType<T>::Type,T>
{
  cuNDA_pshrinkd( typename realType<T>::Type _gamma,typename realType<T>::Type _p ) : gamma(_gamma), p(_p) {}
  __device__ T operator()(const T &x, const typename realType<T>::Type &s) const {
    return x/s*max(s-gamma*pow(s,p-1),typename realType<T>::Type(0));
  }
  typename realType<T>::Type gamma;
  typename realType<T>::Type p;
};

template<class T> void
Gadgetron::pshrinkd( cuNDArray<T> *x, cuNDArray<typename realType<T>::Type> *s, typename realType<T>::Type gamma,typename realType<T>::Type p, cuNDArray<T> *out )
{
  if( x == 0x0 )
    throw std::runtime_error("Gadgetron::shrinkd(): Invalid input array");

  thrust::device_ptr<T> xPtr = x->get_device_ptr();
  thrust::device_ptr<T> outPtr = (out == 0x0) ? x->get_device_ptr() : out->get_device_ptr();
  thrust::device_ptr<typename realType<T>::Type> sPtr = s->get_device_ptr();
  thrust::transform(xPtr,xPtr+x->get_number_of_elements(),sPtr,outPtr,cuNDA_pshrinkd<T>(gamma,p));
}

//
// Instantiation
//

template boost::shared_ptr< cuNDArray<float> > Gadgetron::abs<float>( const cuNDArray<float>* );
template void Gadgetron::abs_inplace<float>( cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::abs_square<float>( const cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::sqrt<float>( const cuNDArray<float>* );
template void Gadgetron::sqrt_inplace<float>( cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::square<float>( const cuNDArray<float>* );
template void Gadgetron::square_inplace<float>( cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::reciprocal<float>( const cuNDArray<float>* );
template void Gadgetron::reciprocal_inplace<float>( cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::reciprocal_sqrt<float>( const cuNDArray<float>* );
template void Gadgetron::reciprocal_sqrt_inplace<float>( cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::sgn<float>( const cuNDArray<float>* );
template void Gadgetron::sgn_inplace<float>( cuNDArray<float>* );
template void Gadgetron::clear<float>( cuNDArray<float>* );
template void Gadgetron::fill<float>( cuNDArray<float>*, float );
template void Gadgetron::clamp<float>( cuNDArray<float>*, float, float );
template void Gadgetron::clamp<float>( cuNDArray<float>*, float, float, float,float );
template void Gadgetron::clamp_min<float>( cuNDArray<float>*, float );
template void Gadgetron::clamp_max<float>( cuNDArray<float>*, float );
template void Gadgetron::normalize<float>( cuNDArray<float>*, float );
template void Gadgetron::shrink1<float>( cuNDArray<float>*, float, cuNDArray<float>* );
template void Gadgetron::pshrink<float>( cuNDArray<float>*, float,float, cuNDArray<float>* );
template void Gadgetron::shrinkd<float> ( cuNDArray<float>*, cuNDArray<float>*, float, cuNDArray<float>* );
template void Gadgetron::pshrinkd<float> ( cuNDArray<float>*, cuNDArray<float>*, float,float, cuNDArray<float>* );

template boost::shared_ptr< cuNDArray<double> > Gadgetron::abs<double>( const cuNDArray<double>* );
template void Gadgetron::abs_inplace<double>( cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::abs_square<double>( const cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::sqrt<double>( const cuNDArray<double>* );
template void Gadgetron::sqrt_inplace<double>( cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::square<double>( const cuNDArray<double>* );
template void Gadgetron::square_inplace<double>( cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::reciprocal<double>( const cuNDArray<double>* );
template void Gadgetron::reciprocal_inplace<double>( cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::reciprocal_sqrt<double>( const cuNDArray<double>* );
template void Gadgetron::reciprocal_sqrt_inplace<double>( cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::sgn<double>( const cuNDArray<double>* );
template void Gadgetron::sgn_inplace<double>( cuNDArray<double>* );
template void Gadgetron::clear<double>( cuNDArray<double>* );
template void Gadgetron::fill<double>( cuNDArray<double>*, double );
template void Gadgetron::clamp<double>( cuNDArray<double>*, double, double );
template void Gadgetron::clamp<double>( cuNDArray<double>*, double, double, double, double );
template void Gadgetron::clamp_min<double>( cuNDArray<double>*, double );
template void Gadgetron::clamp_max<double>( cuNDArray<double>*, double );
template void Gadgetron::normalize<double>( cuNDArray<double>*, double );
template void Gadgetron::shrink1<double>( cuNDArray<double>*, double, cuNDArray<double>* );
template void Gadgetron::pshrink<double>( cuNDArray<double>*, double,double, cuNDArray<double>* );
template void Gadgetron::shrinkd<double> ( cuNDArray<double>*, cuNDArray<double>*, double, cuNDArray<double>* );
template void Gadgetron::pshrinkd<double> ( cuNDArray<double>*, cuNDArray<double>*, double,double, cuNDArray<double>* );


template void Gadgetron::fill<bool>( cuNDArray<bool>*, bool );
/*template boost::shared_ptr< cuNDArray<float> > Gadgetron::abs< std::complex<float> >( const cuNDArray< std::complex<float> >* );
template boost::shared_ptr< cuNDArray< std::complex<float> > > Gadgetron::sqrt< std::complex<float> >( const cuNDArray< std::complex<float> >* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::abs_square< std::complex<float> >( const cuNDArray< std::complex<float> >* );
template void Gadgetron::sqrt_inplace< std::complex<float> >( cuNDArray< std::complex<float> >* );
template boost::shared_ptr< cuNDArray< std::complex<float> > > Gadgetron::square< std::complex<float> >( const cuNDArray< std::complex<float> >* );
template void Gadgetron::square_inplace< std::complex<float> >( cuNDArray< std::complex<float> >* );
template boost::shared_ptr< cuNDArray< std::complex<float> > > Gadgetron::reciprocal< std::complex<float> >( const cuNDArray< std::complex<float> >* );
template void Gadgetron::reciprocal_inplace< std::complex<float> >( cuNDArray< std::complex<float> >* );
template boost::shared_ptr< cuNDArray< std::complex<float> > > Gadgetron::reciprocal_sqrt< std::complex<float> >( const cuNDArray< std::complex<float> >* );
template void Gadgetron::reciprocal_sqrt_inplace< std::complex<float> >( cuNDArray< std::complex<float> >* );
template void Gadgetron::clear< std::complex<float> >( cuNDArray< std::complex<float> >* );
template void Gadgetron::fill< std::complex<float> >( cuNDArray< std::complex<float> >*, std::complex<float> );
template void Gadgetron::normalize< std::complex<float> >( cuNDArray< std::complex<float> >*, float );
template void Gadgetron::shrink1< std::complex<float> >( cuNDArray< std::complex<float> >*, float );
template void Gadgetron::shrinkd< std::complex<float> > ( cuNDArray< std::complex<float> >*, cuNDArray<float>*, float );

template boost::shared_ptr< cuNDArray<double> > Gadgetron::abs< std::complex<double> >( const cuNDArray< std::complex<double> >* );
template boost::shared_ptr< cuNDArray< std::complex<double> > > Gadgetron::sqrt< std::complex<double> >( const cuNDArray< std::complex<double> >* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::abs_square< std::complex<double> >( const cuNDArray< std::complex<double> >* );
template void Gadgetron::sqrt_inplace< std::complex<double> >( cuNDArray< std::complex<double> >* );
template boost::shared_ptr< cuNDArray< std::complex<double> > > Gadgetron::square< std::complex<double> >( const cuNDArray< std::complex<double> >* );
template void Gadgetron::square_inplace< std::complex<double> >( cuNDArray< std::complex<double> >* );
template boost::shared_ptr< cuNDArray< std::complex<double> > > Gadgetron::reciprocal< std::complex<double> >( const cuNDArray< std::complex<double> >* );
template void Gadgetron::reciprocal_inplace< std::complex<double> >( cuNDArray< std::complex<double> >* );
template boost::shared_ptr< cuNDArray< std::complex<double> > > Gadgetron::reciprocal_sqrt< std::complex<double> >( const cuNDArray< std::complex<double> >* );
template void Gadgetron::reciprocal_sqrt_inplace< std::complex<double> >( cuNDArray< std::complex<double> >* );
template void Gadgetron::clear< std::complex<double> >( cuNDArray< std::complex<double> >* );
template void Gadgetron::fill< std::complex<double> >( cuNDArray< std::complex<double> >*, std::complex<double> );
template void Gadgetron::normalize< std::complex<double> >( cuNDArray< std::complex<double> >*, double );
template void Gadgetron::shrink1< std::complex<double> >( cuNDArray< std::complex<double> >*, double );
template void Gadgetron::shrinkd< std::complex<double> > ( cuNDArray< std::complex<double> >*, cuNDArray<double>*, double );
*/
template boost::shared_ptr< cuNDArray<float> > Gadgetron::abs< complext<float> >( const cuNDArray< complext<float> >* );
template void Gadgetron::abs_inplace<complext<float> >(cuNDArray<complext<float> >*);
template boost::shared_ptr< cuNDArray< complext<float> > > Gadgetron::sqrt< complext<float> >( const cuNDArray< complext<float> >* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::abs_square< complext<float> >( const cuNDArray< complext<float> >* );
template void Gadgetron::sqrt_inplace< complext<float> >( cuNDArray< complext<float> >* );
template boost::shared_ptr< cuNDArray< complext<float> > > Gadgetron::square< complext<float> >( const cuNDArray< complext<float> >* );
template void Gadgetron::square_inplace< complext<float> >( cuNDArray< complext<float> >* );
template boost::shared_ptr< cuNDArray< complext<float> > > Gadgetron::reciprocal< complext<float> >( const cuNDArray< complext<float> >* );
template void Gadgetron::reciprocal_inplace< complext<float> >( cuNDArray< complext<float> >* );
template boost::shared_ptr< cuNDArray< complext<float> > > Gadgetron::reciprocal_sqrt< complext<float> >( const cuNDArray< complext<float> >* );
template void Gadgetron::reciprocal_sqrt_inplace< complext<float> >( cuNDArray< complext<float> >* );
template boost::shared_ptr< cuNDArray<complext<float> > > Gadgetron::sgn<complext<float> >( const cuNDArray<complext<float> >* );
template void Gadgetron::sgn_inplace<complext<float> >( cuNDArray<complext<float> >* );
template void Gadgetron::clear< complext<float> >( cuNDArray< complext<float> >* );
template void Gadgetron::fill< complext<float> >( cuNDArray< complext<float> >*, complext<float> );
template void Gadgetron::clamp< complext<float> >( cuNDArray< complext<float> >*, float, float );
template void Gadgetron::clamp_min< complext<float> >( cuNDArray< complext<float> >*, float );
template void Gadgetron::clamp_max< complext< float> >( cuNDArray<complext<float> >*, float );
template void Gadgetron::normalize< complext<float> >( cuNDArray< complext<float> >*, float );
template void Gadgetron::shrink1< complext<float> >( cuNDArray< complext<float> >*, float, cuNDArray< complext<float> >* );
template void Gadgetron::pshrink< complext<float> >( cuNDArray< complext<float> >*, float,float, cuNDArray< complext<float> >* );
template void Gadgetron::shrinkd< complext<float> > ( cuNDArray< complext<float> >*, cuNDArray<float>*, float, cuNDArray< complext<float> >* );
template void Gadgetron::pshrinkd< complext<float> > ( cuNDArray< complext<float> >*, cuNDArray<float>*, float,float, cuNDArray< complext<float> >* );

template boost::shared_ptr< cuNDArray<double> > Gadgetron::abs< complext<double> >( const cuNDArray< complext<double> >* );
template boost::shared_ptr< cuNDArray< complext<double> > > Gadgetron::sqrt< complext<double> >( const cuNDArray< complext<double> >* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::abs_square< complext<double> >( const cuNDArray< complext<double> >* );
template void Gadgetron::sqrt_inplace< complext<double> >( cuNDArray< complext<double> >* );
template boost::shared_ptr< cuNDArray< complext<double> > > Gadgetron::square< complext<double> >( const cuNDArray< complext<double> >* );
template void Gadgetron::square_inplace< complext<double> >( cuNDArray< complext<double> >* );
template boost::shared_ptr< cuNDArray< complext<double> > > Gadgetron::reciprocal< complext<double> >( const cuNDArray< complext<double> >* );
template void Gadgetron::reciprocal_inplace< complext<double> >( cuNDArray< complext<double> >* );
template boost::shared_ptr< cuNDArray< complext<double> > > Gadgetron::reciprocal_sqrt< complext<double> >( const cuNDArray< complext<double> >* );
template void Gadgetron::reciprocal_sqrt_inplace< complext<double> >( cuNDArray< complext<double> >* );
template boost::shared_ptr< cuNDArray<complext<double> > > Gadgetron::sgn<complext<double> >( const cuNDArray<complext<double> >* );
template void Gadgetron::sgn_inplace<complext<double> >( cuNDArray<complext<double> >* );
template void Gadgetron::clear< complext<double> >( cuNDArray< complext<double> >* );
template void Gadgetron::fill< complext<double> >( cuNDArray< complext<double> >*, complext<double> );
template void Gadgetron::clamp< complext<double> >( cuNDArray< complext<double> >*, double, double );
template void Gadgetron::clamp_min< complext<double> >( cuNDArray< complext<double> >*, double );
template void Gadgetron::clamp_max< complext<double> >( cuNDArray<complext<double> >*, double );
template void Gadgetron::normalize< complext<double> >( cuNDArray< complext<double> >*, double );
template void Gadgetron::shrink1< complext<double> >( cuNDArray< complext<double> >*, double, cuNDArray< complext<double> >* );
template void Gadgetron::pshrink< complext<double> >( cuNDArray< complext<double> >*, double, double, cuNDArray< complext<double> >* );
template void Gadgetron::shrinkd< complext<double> > ( cuNDArray< complext<double> >*, cuNDArray<double>*, double, cuNDArray< complext<double> >* );
template void Gadgetron::pshrinkd< complext<double> > ( cuNDArray< complext<double> >*, cuNDArray<double>*, double,double, cuNDArray< complext<double> >* );

template boost::shared_ptr< cuNDArray<float> > Gadgetron::real<float>( const cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::imag<float>( const cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::conj<float>( const cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::real<float_complext>( const cuNDArray<float_complext>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::imag<float_complext>( const cuNDArray<float_complext>* );
template boost::shared_ptr< cuNDArray<float_complext> > Gadgetron::conj<float_complext>( const cuNDArray<float_complext>* );
template boost::shared_ptr< cuNDArray<float_complext> > Gadgetron::real_to_complex<float_complext>( const cuNDArray<float>* );

template boost::shared_ptr< cuNDArray<double> > Gadgetron::real<double>( const cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::imag<double>( const cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::conj<double>( const cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::real<double_complext>( const cuNDArray<double_complext>* );
template boost::shared_ptr< cuNDArray<double> > Gadgetron::imag<double_complext>( const cuNDArray<double_complext>* );
template boost::shared_ptr< cuNDArray<double_complext> > Gadgetron::conj<double_complext>( const cuNDArray<double_complext>* );
template boost::shared_ptr< cuNDArray<double_complext> > Gadgetron::real_to_complex<double_complext>( const cuNDArray<double>* );

template boost::shared_ptr< cuNDArray<double> > Gadgetron::convert_to<float,double>( const cuNDArray<float>* );
template boost::shared_ptr< cuNDArray<float> > Gadgetron::convert_to<double,float>( const cuNDArray<double>* );
template boost::shared_ptr< cuNDArray<double_complext> > Gadgetron::convert_to<float_complext,double_complext>( const cuNDArray<float_complext>* );
template boost::shared_ptr< cuNDArray<float_complext> > Gadgetron::convert_to<double_complext,float_complext>( const cuNDArray<double_complext>* );

template void Gadgetron::convert_to<float,double>( cuNDArray<float>*,cuNDArray<double>* );
template void Gadgetron::convert_to<double,float>( cuNDArray<double>*, cuNDArray<float>* );
template void Gadgetron::convert_to<float_complext,double_complext>( cuNDArray<float_complext>*,cuNDArray<double_complext>*  );
template void Gadgetron::convert_to<double_complext,float_complext>( cuNDArray<double_complext>*, cuNDArray<float_complext>*);
